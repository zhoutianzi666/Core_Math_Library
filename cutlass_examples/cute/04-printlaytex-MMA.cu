#include <hip/hip_runtime.h>
#include <stdlib.h>

#include <cute/tensor.hpp>

using namespace cute;
int main() {
    // mma
    using mma_op = SM80_16x8x16_F32F16F16F32_TN;
    using mma_traits = MMA_Traits<mma_op>;
    using mma_atom = MMA_Atom<mma_traits>;
    static constexpr int kMmaEURepeatM = 1;
    static constexpr int kMmaEURepeatN = 4;
    static constexpr int kMmaEURepeatK = 1;

    using mma_atom_shape = mma_traits::Shape_MNK;
    static constexpr int kMmaPM = 1;
    static constexpr int kMmaPN = 2;
    static constexpr int kMmaPK = 1;
    using MMA_EU_RepeatT = decltype(make_layout(make_shape(
        Int<kMmaEURepeatM>{}, Int<kMmaEURepeatN>{}, Int<kMmaEURepeatK>{})));
    using MMA_P_T = Tile<Int<kMmaPM>, Int<kMmaPN>, Int<kMmaPK>>;
    using MMA = decltype(make_tiled_mma(mma_atom{}, MMA_EU_RepeatT{}, MMA_P_T{}));
    
    auto thr_mma = MMA{}.get_thread_slice(0);
   // print(thr_mma);
   // print(MMA{});
    
   
   print_latex(MMA{});

}
