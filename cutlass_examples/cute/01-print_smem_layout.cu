#include <hip/hip_runtime.h>
#include <stdlib.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

#define PRINT(name, content) \
    print(name);             \
    print(" : ");            \
    print(content);          \
    print("\n");

using namespace cute;
// 
int main() {
    constexpr int M = 64;
    constexpr int N = 64;
    constexpr int N1 = 64;
    
    using SmemLayoutAtom = decltype(composition(
        Swizzle<3, 3, 3>{},
        make_layout(make_shape(Int<8>{}, Int<N1>{}),
                    make_stride(Int<N1>{}, Int<1>{})
                    )));
    using SmemLayout = decltype(tile_to_shape(SmemLayoutAtom{},
                                              make_shape(Int<M>{}, Int<N>{})));

    //PRINT("SmemLayout", SmemLayout{});
    //print_latex(SmemLayout{});
    //print_layout(SmemLayout{});
    
    // git reset --hard v3.1.0 才有下面这句话！
    // print_latex(SmemLayout{}.layout_fn());
    print_layout(SmemLayout{}.layout_fn());
    //PRINT("SmemLayout", SmemLayout{}.layout_fn());
}
