#include <hip/hip_runtime.h>
#include <stdlib.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

#define PRINT(name, content) \
    print(name);             \
    print(" : ");            \
    print(content);          \
    print("\n");

#define PRINTTENSOR(name, content) \
    print(name);                   \
    print(" : ");                  \
    print_tensor(content);         \
    print("\n");
    



using namespace cute;


int main()
{



    using T = cute::half_t;
    // copy from shared memory to register
    // use mma tiled ,so no tiled here
    using s2r_copy_op = SM75_U32x4_LDSM_N;
    using s2r_copy_traits = Copy_Traits<s2r_copy_op>;
    using s2r_copy_atom = Copy_Atom<s2r_copy_traits, T>;
    using S2RCopyAtomA = s2r_copy_atom;
    using S2RCopyAtomB = s2r_copy_atom;

    // mma
    using mma_op = SM80_16x8x16_F16F16F16F16_TN;
    using mma_traits = MMA_Traits<mma_op>;
    using mma_atom = MMA_Atom<mma_traits>;
    static constexpr int kMmaEURepeatM = 2;
    static constexpr int kMmaEURepeatN = 2;
    static constexpr int kMmaEURepeatK = 1;

    using mma_atom_shape = mma_traits::Shape_MNK;
    static constexpr int kMmaPM = 1;
    static constexpr int kMmaPN = 1;
    static constexpr int kMmaPK = 1;
    using MMA_EU_RepeatT = decltype(make_layout(make_shape(
        Int<kMmaEURepeatM>{}, Int<kMmaEURepeatN>{}, Int<kMmaEURepeatK>{})));
    using MMA_P_T = Tile<Int<kMmaPM>, Int<kMmaPN>, Int<kMmaPK>>;
    using MMA = decltype(make_tiled_mma(mma_atom{}, MMA_EU_RepeatT{}, MMA_P_T{}));
    auto s2r_tiled_copy_a = make_tiled_copy_A(S2RCopyAtomA{}, MMA{});
    //auto s2r_tiled_copy_b = make_tiled_copy_B(S2RCopyAtomB{}, MMA{});
    print_latex(s2r_tiled_copy_a);
}
