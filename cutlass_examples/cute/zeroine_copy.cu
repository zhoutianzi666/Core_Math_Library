#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>

#include <cute/tensor.hpp>

#define PRINT(name, content) \
    print(name);             \
    print(" : ");            \
    print(content);          \
    print("\n");

#define PRINTTENSOR(name, content) \
    print(name);                   \
    print(" : ");                  \
    print_tensor(content);         \
    print("\n");
    



using namespace cute;

template <typename T, typename G2SCopy, typename S2RCopy, typename SmemLayout, int M, int N>
__global__ void copy_global_shm_register(const T *Aptr)
{
    int idx = threadIdx.x;
    extern __shared__ T shm_data[];
    T *Ashm = shm_data;

    auto gA = make_tensor(make_gmem_ptr(Aptr), make_shape(Int<M>{}, Int<N>{}), make_stride(Int<N>{}, Int<1>{}));

    auto sA = make_tensor(make_smem_ptr(Ashm), SmemLayout{});

    auto rA = make_tensor_like(gA);

    G2SCopy g2s_tiled_copy;
    auto g2s_thr_copy = g2s_tiled_copy.get_slice(idx);
    auto tAgA = g2s_thr_copy.partition_S(gA);
    auto tAsA = g2s_thr_copy.partition_D(sA);
    cute::copy(g2s_tiled_copy, tAgA, tAsA);

    S2RCopy s2r_tiled_copy;
    auto s2r_thr_copy = s2r_tiled_copy.get_slice(idx);
    // error: In CopyAtom, src layout doesn't vectorize into registers. This src layout is incompatible with this tiled copy.
    // auto stAsA = s2r_thr_copy.partition_S(sA);
    auto stAsA = s2r_thr_copy.retile_S(tAsA);
    auto tArA = s2r_thr_copy.partition_D(rA);
    cute::copy(s2r_tiled_copy, stAsA, tArA);

    if (idx == 0)
    {
        // ((_8,_1),_4,_4)
        // (CPY, CPY_M, CPY_N)
        // 其中CPY由copy_op決定，這裡對2個OP都是128bit=16bytes=8half, 確定了copy的基本操作單位。
        // CPY_M = M / (ThrLayout_M * ValLayout_M), CPY_N = N / (ThrLayout_N * ValLayout_N), CPY_M、CPY_N和TiledCopy共同確定了拷貝的元素數量。

        PRINT("tAgA", tAgA.shape());
        PRINT("tAsA", tAsA.shape());
        PRINT("stAsA", stAsA.shape());
        PRINT("tArA", tArA.shape());
    }
}


int main()
{
    using T = cute::half_t;
    int device;
    hipError_t err = hipGetDevice(&device);
    if (err != hipSuccess) {
        fprintf(stderr, "hipGetDevice returned error %d (%s)\n", err, hipGetErrorString(err));
        return -1;
    }

    int sharedMemPerBlock;
    err = hipDeviceGetAttribute(&sharedMemPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, device);
    if (err != hipSuccess) {
        fprintf(stderr, "hipDeviceGetAttribute returned error %d (%s)\n", err, hipGetErrorString(err));
        return -1;
    }

    printf("Max shared memory per block for device %d is %d bytes\n", device, sharedMemPerBlock);

    // prefer more shm and less L1 cache
    err = hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to set cache config (%s)\n", hipGetErrorString(err));
        exit(-1);
    }

    // constexpr int M = 128;
    // constexpr int N = 32;
    constexpr int M = 128;
    constexpr int N = 128;

    hipEvent_t start, end;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&end);

    // global memory to shared memory copy
    // 利用cp.async完成global memory到shared memory的异步拷贝。
    // 每个线程完成128bit=16bytes的数据拷贝
    using g2s_copy_op = SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>;
    using g2s_copy_traits = Copy_Traits<g2s_copy_op>;
    using g2s_copy_atom = Copy_Atom<g2s_copy_traits, T>;
    /** Produce a TiledCopy from logical thread and values layouts.
     * The thread and value layouts map coordinates to thr_idx and val_idx.
     *    The product of these layouts is taken to produce the TV layout and the Tiler.
     * Useful when threads and values need very specific mappings onto coordinates
     *    in the target tensors.
     *
     *
     * make_tiled_copy(Copy_Atom<Args...> const& copy_atom,
                ThrLayout          const& thr_layout = {},     // (m,n) -> thr_idx
                ValLayout          const& val_layout = {})     // (m,n) -> val_idx
     */
    // (32,4) threads layout, every thread handle 8 elements(8half=16bytes=128bit)
    using G2SCopy =
        decltype(make_tiled_copy(g2s_copy_atom{},
                                 make_layout(make_shape(Int<32>{}, Int<4>{}),
                                             make_stride(Int<4>{}, Int<1>{})),
                                 make_layout(make_shape(Int<1>{}, Int<8>{}))));
    auto a = Swizzle<1, 1, 1>{};

    // logical layout (8,32) to physical layout (8,8,8)
    using SmemLayoutAtom = decltype(composition(
        Swizzle<1, 1, 1>{},
        make_layout(make_shape(Int<8>{}, Int<32>{}),
                    make_stride(Int<32>{}, Int<1>{}))));
    using SmemLayout = decltype(tile_to_shape(SmemLayoutAtom{},
                                              make_shape(Int<M>{}, Int<N>{})));

    static constexpr int shm_size = cute::cosize(SmemLayout{}) * sizeof(T);

    std::cout << shm_size << std::endl;
    exit(0);

    // shared memory to register copy
    /*
    ldmatrix可以实现warp level共享内存到寄存器的数据搬运。
    ldmatrix由于是单线程提供16Byte的数据地址，warp内所有线程可以提供512Byte的数据到寄存器的加载，单指令实现16x16 float16矩阵的加载，减少指令数。
    */
    using s2r_copy_op = SM75_U32x4_LDSM_N;
    using s2r_copy_traits = Copy_Traits<s2r_copy_op>;
    using s2r_copy_atom = Copy_Atom<s2r_copy_traits, T>;
    using S2RCopy =
        decltype(make_tiled_copy(s2r_copy_atom{},
                                 make_layout(make_shape(Int<32>{}, Int<4>{}),
                                             make_stride(Int<4>{}, Int<1>{})),
                                 make_layout(make_shape(Int<1>{}, Int<8>{}))));

    T *Aptr;
    hipMalloc(&Aptr, sizeof(T) * M * N);
    dim3 block(128);
    hipEventRecord(start);
    int count = 100;
    for (int i = 0; i < count; ++i) {
        copy_global_shm_register<T, G2SCopy, S2RCopy, SmemLayout, M, N><<<1, block, shm_size>>>(Aptr);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);
    err = hipGetLastError();
    if (err == hipSuccess) {
        printf("err = %d, str = %s\n", err, hipGetErrorString(err));
    }
    else {
        printf("err = %d, str = %s\n", err, hipGetErrorString(err));
    }
    hipEventElapsedTime(&elapsedTime, start, end);
    std::cout << "copy_global_shm_register took " << elapsedTime / count << "ms." << std::endl;
}
