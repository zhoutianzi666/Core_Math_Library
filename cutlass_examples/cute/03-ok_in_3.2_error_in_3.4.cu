#include <hip/hip_runtime.h>
#include <stdlib.h>

#include <cute/tensor.hpp>
//git reset --hard v3.2.1
//git reset --hard v3.4.1

#define PRINT(name, content) \
    print(name);             \
    print(" : ");            \
    print(content);          \
    print("\n");

#define PRINTTENSOR(name, content) \
    print(name);                   \
    print(" : ");                  \
    print_tensor(content);         \
    print("\n");

using namespace cute;
int main() {
    using T = cute::half_t;
    using s2r_copy_op = SM75_U32x4_LDSM_N;
    using s2r_copy_traits = Copy_Traits<s2r_copy_op>;
    using s2r_copy_atom = Copy_Atom<s2r_copy_traits, T>;
    
    using S2RCopyAtomA = s2r_copy_atom;
    using S2RCopyAtomB = s2r_copy_atom;

    // mma
    using mma_op = SM80_16x8x16_F32F16F16F32_TN;
    using mma_traits = MMA_Traits<mma_op>;
    using mma_atom = MMA_Atom<mma_traits>;
    static constexpr int kMmaEURepeatM = 1;
    static constexpr int kMmaEURepeatN = 1;
    static constexpr int kMmaEURepeatK = 1;

    using mma_atom_shape = mma_traits::Shape_MNK;
    static constexpr int kMmaPM = 1;
    static constexpr int kMmaPN = 1;
    static constexpr int kMmaPK = 2;
    using MMA_EU_RepeatT = decltype(make_layout(make_shape(
        Int<kMmaEURepeatM>{}, Int<kMmaEURepeatN>{}, Int<kMmaEURepeatK>{})));
    using MMA_P_T = Tile<Int<kMmaPM>, Int<kMmaPN>, Int<kMmaPK>>;
    using MMA = decltype(make_tiled_mma(mma_atom{}, MMA_EU_RepeatT{}, MMA_P_T{}));
    auto s2r_tiled_copy_a = make_tiled_copy_A(S2RCopyAtomA{}, MMA{});
    auto s2r_tiled_copy_b = make_tiled_copy_B(S2RCopyAtomB{}, MMA{});

    // 这个打印的是src到dst的关系哦！
    //print_latex(s2r_tiled_copy_a);
    //print_latex(s2r_tiled_copy_b);
    
    
    print_latex(MMA{});
}
