#include "hip/hip_runtime.h"

#include <stdio.h>
#include <assert.h>
#include <chrono>
#include <ctime>
#include <iostream>
#include <ratio>

#include "cutlass/transform/threadblock/predicated_tile_iterator.h"
#include "cutlass/layout/pitch_linear.h"
#include "cutlass/transform/pitch_linear_thread_map.h"

using DATATYPE = int;

template <typename Iterator>
__global__ void copy(
    typename Iterator::Params dst_params,
    typename Iterator::Element *dst_pointer,
    typename Iterator::Params src_params,
    typename Iterator::Element *src_pointer,
    cutlass::Coord<2> extent) {


    Iterator dst_iterator(dst_params, dst_pointer, extent, threadIdx.x);
    Iterator src_iterator(src_params, src_pointer, extent, threadIdx.x);

    // PredicatedTileIterator uses PitchLinear layout and therefore takes in a PitchLinearShape.
    // The contiguous dimension can be accessed via Iterator::Shape::kContiguous and the strided
    // dimension can be accessed via Iterator::Shape::kStrided
    int iterations = (extent[1] + Iterator::Shape::kStrided - 1) / Iterator::Shape::kStrided;

    typename Iterator::Fragment fragment;

    for(int i = 0; i < fragment.size(); ++i) {
      fragment[i] = 0;
    }

    src_iterator.load(fragment);
    dst_iterator.store(fragment);


    ++src_iterator;
    ++dst_iterator;

    for(; iterations > 1; --iterations) {

      src_iterator.load(fragment);
      dst_iterator.store(fragment);

      ++src_iterator;
      ++dst_iterator;
    }
}


int main1(void) {
  int M = 900;
  int N = 60;

  // Note input is in CPU place
  DATATYPE *input;
  int input_size = M * N;

  hipError_t status = hipHostMalloc(&input, sizeof(DATATYPE) * input_size);
  assert(status == hipSuccess);
  //init(input, input_size);

  // dev_input is from weight
  DATATYPE *dev_input, *dev_out;
  hipMalloc((void **)&dev_input, input_size * sizeof(DATATYPE));
  hipMalloc((void **)&dev_out, input_size * sizeof(DATATYPE));
  hipMemcpy(dev_input, input, input_size * sizeof(DATATYPE), hipMemcpyHostToDevice);


  using Shape = cutlass::layout::PitchLinearShape<16, 4>;
  using Layout = cutlass::layout::PitchLinear;
  using Element = int;
  int const kThreads = 32;
  using ThreadMap = cutlass::transform::PitchLinearStripminedThreadMap<Shape, kThreads>;
  using Iterator = cutlass::transform::threadblock::PredicatedTileIterator<Shape, Element, Layout, 1, ThreadMap>;
  typename Iterator::Params dst_params({M});
  typename Iterator::Params src_params({M});

  std::cout << Iterator::TileAccessIterator::UnderlyingPredicates::kAccessesPerVector << std::endl;
  std::cout << Iterator::TileAccessIterator::UnderlyingPredicates::kPredicateWordCount << std::endl;

  
  dim3 block(kThreads, 1);
  dim3 grid(1, 1);
  copy<Iterator><<< grid, block >>>(
          dst_params,
          dev_out,
          src_params,
          dev_input,
          cutlass::make_Coord(M, N));




  hipDeviceReset();
  hipHostFree(input);
  hipFree(dev_input);
  hipFree(dev_out);
  return 0;
}
