
#include <hip/hip_runtime.h>
#include <cstdio>
#include <hip/hip_fp16.h>
#define dtype float
#define flops_each_thread 40000
// nvcc cuda_core_fp32.cu -arch sm_80 -o a.out
// cuobjdump -ptx a.out
// 我测试出来的 A100 cuda core fp32 flops 17.870323 TFlops和宣传的
__global__ void kernel(const dtype *x, dtype *y, int N) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = global_idx; i < N; i += blockDim.x * gridDim.x) {
        dtype a = x[i];
        dtype b;
        
        for (int ii = 0; ii < flops_each_thread; ii++) {
            b = b + a;
            b = b * b;
        }

        y[i] = b;
    }
}

int main() {
    const int WARMUP_ITER = 10;
    const int BENCH_ITER = 50;
    const int N_DATA = 1024 * 1024 * 128;
    dtype *x, *y;
    hipMalloc(&x, N_DATA * sizeof(dtype));
    hipMalloc(&y, N_DATA * sizeof(dtype));
    hipMemset(x, 0, N_DATA * sizeof(dtype));
    int blocksize = 32;
    int grid = 1024*1024;
    hipEvent_t start, stop;

    for (int i = 0; i < WARMUP_ITER; ++i) {
        kernel<<<grid, blocksize>>>(x, y, N_DATA);
    }

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int i = 0; i < BENCH_ITER ; ++i) {
        kernel<<<grid, blocksize>>>(x, y, N_DATA);
    }
    hipEventRecord(stop);

    float time_ms = 0.f;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_ms, start, stop);
    printf("time: %f ms\n", time_ms);
    
    float flops = (N_DATA * (float)(flops_each_thread) * 2 * BENCH_ITER) / (time_ms) * 1000;
    printf("flops %f TFlops\n", flops/1000000000000);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(x);
    hipFree(y);
    return 0;
}