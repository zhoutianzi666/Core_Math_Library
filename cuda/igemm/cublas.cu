#include <algorithm>
#include <iostream>

#include "hipblas.h"
#include "utility.h"



void CublasIgemmNN(hipblasHandle_t& handle, int M, int N, int K,
                           DATATYPE const *A, int lda, 
                           DATATYPE const *B,int ldb, 
                           int32_t const *bias,
                           C_DATATYPE *C, int ldc) {

const int alpha = 1;
const int beta = 0;

hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
               N, M, K, &alpha,
               B, HIP_R_8I, N,
               A, HIP_R_8I, K,
               &beta,
              C, HIP_R_32I, N,
              HIPBLAS_COMPUTE_32I,
              CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

