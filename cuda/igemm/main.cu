#include "hip/hip_runtime.h"

#include <stdio.h>

#include <chrono>
#include <ctime>
#include <iostream>
#include <ratio>

#include "hipblas.h"
#include "utility.h"

#define WARMUP 10
#define REPEATE 10

void CUDA_CHECK(hipError_t status) {
  if (status != hipSuccess) {
    printf("分配paged内存失败\n");
  }
}

int main(void) {
  int m = 512;
  int n = 512;
  int k = 512;

  DATATYPE *a, *b;
  BIAS_DATATYPE *bias;

  hipError_t status = hipHostMalloc(&a, sizeof(DATATYPE) * m * k);
  CUDA_CHECK(status);
  status = hipHostMalloc(&b, sizeof(DATATYPE) * k * n);
  CUDA_CHECK(status);
  status = hipHostMalloc(&bias, sizeof(BIAS_DATATYPE) * n);
  CUDA_CHECK(status);

  init(a, m * k);
  init(b, k * n);
  init(bias, k * n);

  C_DATATYPE *c;
  hipHostMalloc(&c, sizeof(C_DATATYPE) * m * n);
  memset(c, 0, sizeof(C_DATATYPE) * m * n);

  DATATYPE *dev_a, *dev_b;
  BIAS_DATATYPE *dev_bias;
  C_DATATYPE *dev_c;
  
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  double time1 = (double)clock() / CLOCKS_PER_SEC;
  using std::chrono::system_clock;
  system_clock::time_point today = system_clock::now();
  
 // allocate the memory on the GPU and copy a and b to GPU
  hipMalloc((void **)&dev_a, m * k * sizeof(DATATYPE));
  hipMalloc((void **)&dev_b, k * n * sizeof(DATATYPE));
  hipMalloc((void **)&dev_bias, n * sizeof(BIAS_DATATYPE));
  hipMalloc((void **)&dev_c, m * n * sizeof(C_DATATYPE));

  hipMemcpy(dev_a, a, m * k * sizeof(DATATYPE), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, k * n * sizeof(DATATYPE), hipMemcpyHostToDevice);
  hipMemcpy(dev_bias, bias, n * sizeof(BIAS_DATATYPE), hipMemcpyHostToDevice);

  for (int i = 0; i < WARMUP; i++) {
    CutlassIgemmNN(n, m, k, dev_a, k, dev_b, k, dev_bias, dev_c, n);
  }

  hipEvent_t beg, end;
  hipEventCreate(&beg);
  hipEventCreate(&end);
  hipEventRecord(beg);

  for (int i = 0; i < REPEATE; i++) {
    CutlassIgemmNN(n, m, k, dev_a, k, dev_b, k, dev_bias, dev_c, n);
  }

  hipEventRecord(end);
  hipEventSynchronize(end);
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, beg, end);
  printf("gpu gemm compute time: %f\n", elapsed_time);
  double Gflops = REPEATE * ((float)m * n * k * 2 / 1000000) / elapsed_time;
  printf("Gflops: %5.2f \n", Gflops);

  hipMemcpy(c, dev_c, m * n * sizeof(C_DATATYPE), hipMemcpyDeviceToHost);

  double time2 = (double)clock() / CLOCKS_PER_SEC;
  system_clock::time_point now = system_clock::now();
  auto ts = std::chrono::duration_cast<std::chrono::microseconds>(now - today);
  std::cout << "gpu total time:" << ts.count() / 1000.0 << "ms" << std::endl;
  printf("gpu total time:%lf\n", double(time2 - time1) * 1000);

  time1 = (double)clock() / CLOCKS_PER_SEC;
  today = system_clock::now();

  int32_t *c_cpu_int32 = (int32_t *)malloc(sizeof(float) * m * n);
  memset(c_cpu_int32, 0, sizeof(int32_t) * m * n);
  naive_gemm_cpu(a, b, c_cpu_int32, m, n, k);

  time2 = (double)clock() / CLOCKS_PER_SEC;
  now = system_clock::now();
  ts = std::chrono::duration_cast<std::chrono::microseconds>(now - today);
  std::cout << "cpu time:" << ts.count() / 1000.0 << "ms" << std::endl;
  printf("cpu time:%lf\n", double(time2 - time1) * 1000);

  printf("max_diff: %d\n", diff(c, c_cpu_int32, m * n));

  hipDeviceReset();
  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);
  free(c_cpu_int32);
  return 0;
}
