#include "hip/hip_runtime.h"

#include <stdio.h>

#include <chrono>
#include <ctime>
#include <iostream>
#include <ratio>

#include "hipblas.h"
#include "utility.h"

#define WARMUP 10
#define REPEATE 10

void CUDA_CHECK(hipError_t status) {
  if (status != hipSuccess) {
    printf("分配paged内存失败\n");
  }
}

int main(void) {
  int m = 512;
  int n = 512;
  int k = 512;

  DATATYPE *a, *b;
  BROADCAST_DATATYPE *broadcast;

  hipError_t status = hipHostMalloc(&a, sizeof(DATATYPE) * m * k);
  CUDA_CHECK(status);
  status = hipHostMalloc(&b, sizeof(DATATYPE) * k * n);
  CUDA_CHECK(status);
  status = hipHostMalloc(&broadcast, sizeof(BROADCAST_DATATYPE) * n);
  CUDA_CHECK(status);

  init(a, m * k);
  init(b, k * n);
  init(broadcast, n);

  C_DATATYPE *c;
  hipHostMalloc(&c, sizeof(C_DATATYPE) * m * n);
  memset(c, 0, sizeof(C_DATATYPE) * m * n);

  DATATYPE *dev_a, *dev_b;
  BROADCAST_DATATYPE *dev_broadcast;
  C_DATATYPE *dev_c;
  
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  double time1 = (double)clock() / CLOCKS_PER_SEC;
  using std::chrono::system_clock;
  system_clock::time_point today = system_clock::now();
  
 // allocate the memory on the GPU and copy a and b to GPU
  hipMalloc((void **)&dev_a, m * k * sizeof(DATATYPE));
  hipMalloc((void **)&dev_b, k * n * sizeof(DATATYPE));
  hipMalloc((void **)&dev_broadcast, n * sizeof(BROADCAST_DATATYPE));
  hipMalloc((void **)&dev_c, m * n * sizeof(C_DATATYPE));

  hipMemcpy(dev_a, a, m * k * sizeof(DATATYPE), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, k * n * sizeof(DATATYPE), hipMemcpyHostToDevice);
  hipMemcpy(dev_broadcast, broadcast, n * sizeof(BROADCAST_DATATYPE), hipMemcpyHostToDevice);

  hipEvent_t beg, end;
  hipEventCreate(&beg);
  hipEventCreate(&end);
  hipEventRecord(beg);

  for (int i = 0; i < WARMUP + REPEATE; i++) {
    const float alpha = 1.0f;
    const float beta = 0.0f;

    if (i == WARMUP) {
      hipEventCreate(&beg);
      hipEventCreate(&end);
      hipEventRecord(beg);
    }
    CutlassIgemmNN(n, m, k, dev_a, k, dev_b, k, dev_broadcast, dev_c, n);
  }

  hipEventRecord(end);
  hipEventSynchronize(end);
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, beg, end);
  printf("gpu gemm compute time: %f\n", elapsed_time);
  double Gflops = REPEATE * ((float)m * n * k * 2 / 1000000) / elapsed_time;
  printf("Gflops: %5.2f \n", Gflops);

  hipMemcpy(c, dev_c, m * n * sizeof(C_DATATYPE), hipMemcpyDeviceToHost);

  double time2 = (double)clock() / CLOCKS_PER_SEC;
  system_clock::time_point now = system_clock::now();
  auto ts = std::chrono::duration_cast<std::chrono::microseconds>(now - today);
  std::cout << "gpu total time:" << ts.count() / 1000.0 << "ms" << std::endl;
  printf("gpu total time:%lf\n", double(time2 - time1) * 1000);

  time1 = (double)clock() / CLOCKS_PER_SEC;
  today = system_clock::now();
  

  // 这个是CPU上的baseline的数据类型！
  // 输出要么是fp32，要么是int32！
  using C_base_DATATYPE = float;
  C_base_DATATYPE *c_cpu_32 = (C_base_DATATYPE *)malloc(sizeof(C_base_DATATYPE) * m * n);
  memset(c_cpu_32, 0, sizeof(C_base_DATATYPE) * m * n);
  naive_gemm_cpu(a, b, c_cpu_32, m, n, k, broadcast);

  time2 = (double)clock() / CLOCKS_PER_SEC;
  now = system_clock::now();
  ts = std::chrono::duration_cast<std::chrono::microseconds>(now - today);
  std::cout << "cpu time:" << ts.count() / 1000.0 << "ms" << std::endl;
  printf("cpu time:%lf\n", double(time2 - time1) * 1000);
  
  std::cout << "max_diff:" << diff(c, c_cpu_32, m * n)  << std::endl;

  hipDeviceReset();
  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);
  hipHostFree(broadcast);
  free(c_cpu_32);
  return 0;
}
