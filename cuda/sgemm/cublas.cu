#include <stdio.h>

#include <chrono>
#include <ctime>
#include <iostream>
#include <ratio>

#include "hipblas.h"
#include "utility.h"

using DATATYPE = float;

void cublas_matmul(hipblasHandle_t& handle, DATATYPE *dev_a, DATATYPE *dev_b, DATATYPE *dev_c, int m,
                      int n, int k) {
const float alpha = 1.0f;
const float beta = 0.0f;
hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, dev_b, n, dev_a, k, &beta, dev_c, n);

hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, 
                        m, k, 
                        &alpha,  dev_b, 
                            HIP_R_32F, 
                            n, 
                            dev_a, 
                            HIP_R_32F, 
                            k, 
                            &beta, 
                            dev_c, 
                            HIP_R_32F, 
                            n, 
                            HIP_R_32F, 
                            CUBLAS_GEMM_DEFAULT_TENSOR_OP);

}
