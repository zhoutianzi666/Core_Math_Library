#include "hip/hip_runtime.h"

#include <stdio.h>
#include <assert.h>

#include <chrono>
#include <ctime>
#include <iostream>
#include <ratio>

#include "hipblas.h"
#include "utility.h"

#define WARMUP 0
#define REPEATE 1

using DATATYPE = float;
using C_DATATYPE = float;

int main(void) {
  int m = 5;
  int n = 5;
  int k = 9; 
  hipSetDevice(3);

  DATATYPE *a, *b;
  a = (DATATYPE *)malloc(sizeof(DATATYPE) * m * k);
  b = (DATATYPE *)malloc(sizeof(DATATYPE) * k * n);
  assert(a);
  assert(b);
  init(a, m * k);
  init(b, k * n);

for (int i = 0; i < m * k ;i++)
{
  a[i] = i;
}

for (int i = 0; i < k * n;i++)
{
  b[i] = i;
}

  C_DATATYPE *c;
  c = (C_DATATYPE *)malloc(sizeof(C_DATATYPE) * m * n);
  assert(c);
  memset(c, 0, sizeof(C_DATATYPE) * m * n);

  DATATYPE *dev_a, *dev_b;
  C_DATATYPE *dev_c;

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // allocate the memory on the GPU
  double time1 = (double)clock() / CLOCKS_PER_SEC;
  using std::chrono::system_clock;
  system_clock::time_point today = system_clock::now();

  hipMalloc((void **)&dev_a, m * k * sizeof(DATATYPE));
  hipMalloc((void **)&dev_b, k * n * sizeof(DATATYPE));
  hipMalloc((void **)&dev_c, m * n * sizeof(C_DATATYPE));

  hipMemcpy(dev_a, a, m * k * sizeof(DATATYPE), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, k * n * sizeof(DATATYPE), hipMemcpyHostToDevice);

  for (int i = 0; i < WARMUP; i++) {
    const float alpha = 1.0f;
    const float beta = 0.0f;
    CutlassSgemmNN(n, m, k, alpha, dev_b, n, dev_a, k, beta, dev_c, n);
    // cublas_matmul(handle, dev_a, dev_b, dev_c, m, n , k);
    // matmul_gpu(dev_a, dev_b, dev_c, m, n, k);
    // matmul_gpu_megengine(dev_a, dev_b, dev_c, m, n, k);
    // matmul_gpu_naive_block(dev_a, dev_b, dev_c, m, n, k);
    // matmul_gpu_naive_block_combine_access(dev_a, dev_b, dev_c, m, n, k);
    //matmul_gpu_naive(dev_a, dev_b, dev_c, m, n, k);
  }

  hipEvent_t beg, end;
  hipEventCreate(&beg);
  hipEventCreate(&end);
  hipEventRecord(beg);

  for (int i = 0; i < REPEATE; i++) {
    const float alpha = 1.0f;
    const float beta = 0.0f;
    CutlassSgemmNN(n, m, k, alpha, dev_b, n, dev_a, k, beta, dev_c, n);
    // cublas_matmul(handle, dev_a, dev_b, dev_c, m, n , k);
    // hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, dev_b, n,
    //             dev_a, k, &beta, dev_c, n);
    // matmul_gpu(dev_a, dev_b, dev_c, m, n, k);
    // matmul_gpu_megengine(dev_a, dev_b, dev_c, m, n, k);
    // matmul_gpu_naive_block(dev_a, dev_b, dev_c, m, n, k);
    // matmul_gpu_naive_block_combine_access(dev_a, dev_b, dev_c, m, n, k);
    //matmul_gpu_naive(dev_a, dev_b, dev_c, m, n, k);
  }

  hipEventRecord(end);
  hipEventSynchronize(end);
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, beg, end);
  
  printf("gpu gemm compute time: %f\n", elapsed_time);
  double Gflops = REPEATE * ((float)m * n * k * 2 / 1000000) / elapsed_time;
  printf("Gflops: %5.2f \n", Gflops);

  hipMemcpy(c, dev_c, m * n * sizeof(C_DATATYPE), hipMemcpyDeviceToHost);

  double time2 = (double)clock() / CLOCKS_PER_SEC;
  system_clock::time_point now = system_clock::now();
  auto ts = std::chrono::duration_cast<std::chrono::microseconds>(now - today);
  std::cout << "gpu total time:" << ts.count() / 1000.0 << "ms" << std::endl;
  printf("gpu total time:%lf\n", double(time2 - time1) * 1000);

  time1 = (double)clock() / CLOCKS_PER_SEC;
  today = system_clock::now();

  float *c_cpu_fp32 = (float *)malloc(sizeof(float) * m * n);
  memset(c_cpu_fp32, 0, sizeof(float) * m * n);
  naive_gemm_cpu(a, b, c_cpu_fp32, m, n, k);

  time2 = (double)clock() / CLOCKS_PER_SEC;
  now = system_clock::now();
  ts = std::chrono::duration_cast<std::chrono::microseconds>(now - today);
  std::cout << "cpu time:" << ts.count() / 1000.0 << "ms" << std::endl;
  printf("cpu time:%lf\n", double(time2 - time1) * 1000);

  printf("max_diff: %f\n", diff(c, c_cpu_fp32, m * n));

  hipDeviceReset();
  free(a);
  free(b);
  free(c);
  free(c_cpu_fp32);
  return 0;
}
