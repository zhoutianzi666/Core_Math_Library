#include <stdio.h>

#include <chrono>
#include <ctime>
#include <iostream>
#include <ratio>

#include "hipblas.h"
#include "hip/hip_runtime.h"


#define WARMUP 10
#define REPEATE 10

using DATATYPE = float;
using ACCU_DATATYPE = float;
#define DATATYPE_BYTE 4
#define ACCU_DATATYPE_BYTE 4

#define BLOCK_X 16
#define BLOCK_Y 16

#define TILE_X 128
#define TILE_X_4 32
#define TILE_Y 128
#define TILE_Y_4 32

#define TILE_K 16

#define WPTN 8
#define WPTM 8
#define WPTN_4 2

void init(DATATYPE *a, int size) {
  for (int i = 0; i < size; i++) {
#if DATATYPE_BYTE == 4
    a[i] = (rand() % 9999) / 10000.0;
#else
    a[i] = __float2half((rand() % 9999) / 10000.0 - 0.5);
#endif
  }
}

__global__ void gemm_kernel_NN(const float *__restrict__ A,
                               const float *__restrict__ B,
                               float4 *__restrict__ C, float alpha, float beta,
                               int M, int N, int K) {
  __shared__ float4 smem_a[2][TILE_K * TILE_Y_4];
  __shared__ float4 smem_b[2][TILE_K * TILE_X_4];

  int tx = threadIdx.x % 16;
  int ty = threadIdx.x / 16;

  int tx4 = threadIdx.x % 4;
  int ty4 = threadIdx.x / 4;

  int tx32 = threadIdx.x % 32;
  int ty32 = threadIdx.x / 32;

  const float *pA = (A + K * TILE_Y * blockIdx.y + ty4 * K + tx4 * 4);
  const float *pB = (B + TILE_X * blockIdx.x + ty32 * N + tx32 * 4);
  float4 *pC = C + TILE_Y * blockIdx.y * N / 4 + TILE_X_4 * blockIdx.x;

  int sts_a_offset = tx4 * 4 * TILE_Y + ty4;
  int sts_b_offset = ty32 * TILE_X_4 + tx32;

  float4 f4_zero = make_float4(0.f, 0.f, 0.f, 0.f);
  bool valid_ld_a_0 = ((blockIdx.y * TILE_Y + ty4) < M) && ((tx4 * 4) < K);
  bool valid_ld_a_1 = ((blockIdx.y * TILE_Y + ty4 + 64) < M) && ((tx4 * 4) < K);
  bool valid_ld_b_0 = ((blockIdx.x * TILE_X + tx32 * 4) < N) && (ty32 < K);
  bool valid_ld_b_1 =
      ((blockIdx.x * TILE_X + tx32 * 4) < N) && ((ty32 + 8) < K);

  float4 ldg_a_reg[2];
  float4 ldg_b_reg[2];

  ldg_a_reg[0] = valid_ld_a_0 ? *(const float4 *)pA : f4_zero;
  ldg_a_reg[1] = valid_ld_a_1 ? *(const float4 *)(pA + 64 * K) : f4_zero;
  ldg_b_reg[0] = valid_ld_b_0 ? *(const float4 *)(pB + 0 * N) : f4_zero;
  ldg_b_reg[1] = valid_ld_b_1 ? *(const float4 *)(pB + 8 * N) : f4_zero;

  float4 c[WPTM][WPTN_4] = {{f4_zero}};

  *((float *)&smem_a[0][0] + sts_a_offset + 0 * TILE_Y + 0) = ldg_a_reg[0].x;
  *((float *)&smem_a[0][0] + sts_a_offset + 1 * TILE_Y + 0) = ldg_a_reg[0].y;
  *((float *)&smem_a[0][0] + sts_a_offset + 2 * TILE_Y + 0) = ldg_a_reg[0].z;
  *((float *)&smem_a[0][0] + sts_a_offset + 3 * TILE_Y + 0) = ldg_a_reg[0].w;
  *((float *)&smem_a[0][0] + sts_a_offset + 0 * TILE_Y + 64) = ldg_a_reg[1].x;
  *((float *)&smem_a[0][0] + sts_a_offset + 1 * TILE_Y + 64) = ldg_a_reg[1].y;
  *((float *)&smem_a[0][0] + sts_a_offset + 2 * TILE_Y + 64) = ldg_a_reg[1].z;
  *((float *)&smem_a[0][0] + sts_a_offset + 3 * TILE_Y + 64) = ldg_a_reg[1].w;

  smem_b[0][sts_b_offset + 0] = ldg_b_reg[0];
  smem_b[0][sts_b_offset + 8 * TILE_X_4] = ldg_b_reg[1];

  __syncthreads();

  int i = 0;
  int write_stage_idx = 1;

  float4 reg_a[2][2];
  float4 reg_b[2][2];

  reg_a[0][0] = smem_a[0][0 + ty];
  reg_a[0][1] = smem_a[0][16 + ty];
  reg_b[0][0] = smem_b[0][0 + tx];
  reg_b[0][1] = smem_b[0][16 + tx];

  do {
    i += 16;
    valid_ld_a_0 = (valid_ld_a_0 && ((tx4 * 4 + i) < K));
    valid_ld_a_1 = (valid_ld_a_1 && ((tx4 * 4 + i) < K));
    valid_ld_b_0 = (valid_ld_b_0 && ((ty32 + i) < K));
    valid_ld_b_1 = (valid_ld_b_1 && ((ty32 + 8 + i) < K));

    ldg_a_reg[0] = (valid_ld_a_0) ? *(const float4 *)(pA + i + 0) : f4_zero;
    ldg_a_reg[1] =
        (valid_ld_a_1) ? *(const float4 *)(pA + i + 64 * K) : f4_zero;
    ldg_b_reg[0] =
        (valid_ld_b_0) ? *(const float4 *)(pB + (i + 0) * N) : f4_zero;
    ldg_b_reg[1] =
        (valid_ld_b_1) ? *(const float4 *)(pB + (i + 8) * N) : f4_zero;

    int load_stage_idx = write_stage_idx ^ 1;

#pragma unroll
    for (int j = 0; j < TILE_K - 1; j++) {
      reg_a[(j + 1) % 2][0] =
          smem_a[load_stage_idx][(j + 1) * TILE_Y_4 + 0 + ty];
      reg_a[(j + 1) % 2][1] =
          smem_a[load_stage_idx][(j + 1) * TILE_Y_4 + 16 + ty];
      reg_b[(j + 1) % 2][0] =
          smem_b[load_stage_idx][(j + 1) * TILE_X_4 + 0 + tx];
      reg_b[(j + 1) % 2][1] =
          smem_b[load_stage_idx][(j + 1) * TILE_X_4 + 16 + tx];
      c[0][0].x += reg_a[j % 2][0].x * reg_b[j % 2][0].x;
      c[0][0].y += reg_a[j % 2][0].x * reg_b[j % 2][0].y;
      c[0][0].z += reg_a[j % 2][0].x * reg_b[j % 2][0].z;
      c[0][0].w += reg_a[j % 2][0].x * reg_b[j % 2][0].w;
      c[0][1].x += reg_a[j % 2][0].x * reg_b[j % 2][1].x;
      c[0][1].y += reg_a[j % 2][0].x * reg_b[j % 2][1].y;
      c[0][1].z += reg_a[j % 2][0].x * reg_b[j % 2][1].z;
      c[0][1].w += reg_a[j % 2][0].x * reg_b[j % 2][1].w;
      c[1][0].x += reg_a[j % 2][0].y * reg_b[j % 2][0].x;
      c[1][0].y += reg_a[j % 2][0].y * reg_b[j % 2][0].y;
      c[1][0].z += reg_a[j % 2][0].y * reg_b[j % 2][0].z;
      c[1][0].w += reg_a[j % 2][0].y * reg_b[j % 2][0].w;
      c[1][1].x += reg_a[j % 2][0].y * reg_b[j % 2][1].x;
      c[1][1].y += reg_a[j % 2][0].y * reg_b[j % 2][1].y;
      c[1][1].z += reg_a[j % 2][0].y * reg_b[j % 2][1].z;
      c[1][1].w += reg_a[j % 2][0].y * reg_b[j % 2][1].w;
      c[2][0].x += reg_a[j % 2][0].z * reg_b[j % 2][0].x;
      c[2][0].y += reg_a[j % 2][0].z * reg_b[j % 2][0].y;
      c[2][0].z += reg_a[j % 2][0].z * reg_b[j % 2][0].z;
      c[2][0].w += reg_a[j % 2][0].z * reg_b[j % 2][0].w;
      c[2][1].x += reg_a[j % 2][0].z * reg_b[j % 2][1].x;
      c[2][1].y += reg_a[j % 2][0].z * reg_b[j % 2][1].y;
      c[2][1].z += reg_a[j % 2][0].z * reg_b[j % 2][1].z;
      c[2][1].w += reg_a[j % 2][0].z * reg_b[j % 2][1].w;
      c[3][0].x += reg_a[j % 2][0].w * reg_b[j % 2][0].x;
      c[3][0].y += reg_a[j % 2][0].w * reg_b[j % 2][0].y;
      c[3][0].z += reg_a[j % 2][0].w * reg_b[j % 2][0].z;
      c[3][0].w += reg_a[j % 2][0].w * reg_b[j % 2][0].w;
      c[3][1].x += reg_a[j % 2][0].w * reg_b[j % 2][1].x;
      c[3][1].y += reg_a[j % 2][0].w * reg_b[j % 2][1].y;
      c[3][1].z += reg_a[j % 2][0].w * reg_b[j % 2][1].z;
      c[3][1].w += reg_a[j % 2][0].w * reg_b[j % 2][1].w;
      c[4][0].x += reg_a[j % 2][1].x * reg_b[j % 2][0].x;
      c[4][0].y += reg_a[j % 2][1].x * reg_b[j % 2][0].y;
      c[4][0].z += reg_a[j % 2][1].x * reg_b[j % 2][0].z;
      c[4][0].w += reg_a[j % 2][1].x * reg_b[j % 2][0].w;
      c[4][1].x += reg_a[j % 2][1].x * reg_b[j % 2][1].x;
      c[4][1].y += reg_a[j % 2][1].x * reg_b[j % 2][1].y;
      c[4][1].z += reg_a[j % 2][1].x * reg_b[j % 2][1].z;
      c[4][1].w += reg_a[j % 2][1].x * reg_b[j % 2][1].w;
      c[5][0].x += reg_a[j % 2][1].y * reg_b[j % 2][0].x;
      c[5][0].y += reg_a[j % 2][1].y * reg_b[j % 2][0].y;
      c[5][0].z += reg_a[j % 2][1].y * reg_b[j % 2][0].z;
      c[5][0].w += reg_a[j % 2][1].y * reg_b[j % 2][0].w;
      c[5][1].x += reg_a[j % 2][1].y * reg_b[j % 2][1].x;
      c[5][1].y += reg_a[j % 2][1].y * reg_b[j % 2][1].y;
      c[5][1].z += reg_a[j % 2][1].y * reg_b[j % 2][1].z;
      c[5][1].w += reg_a[j % 2][1].y * reg_b[j % 2][1].w;
      c[6][0].x += reg_a[j % 2][1].z * reg_b[j % 2][0].x;
      c[6][0].y += reg_a[j % 2][1].z * reg_b[j % 2][0].y;
      c[6][0].z += reg_a[j % 2][1].z * reg_b[j % 2][0].z;
      c[6][0].w += reg_a[j % 2][1].z * reg_b[j % 2][0].w;
      c[6][1].x += reg_a[j % 2][1].z * reg_b[j % 2][1].x;
      c[6][1].y += reg_a[j % 2][1].z * reg_b[j % 2][1].y;
      c[6][1].z += reg_a[j % 2][1].z * reg_b[j % 2][1].z;
      c[6][1].w += reg_a[j % 2][1].z * reg_b[j % 2][1].w;
      c[7][0].x += reg_a[j % 2][1].w * reg_b[j % 2][0].x;
      c[7][0].y += reg_a[j % 2][1].w * reg_b[j % 2][0].y;
      c[7][0].z += reg_a[j % 2][1].w * reg_b[j % 2][0].z;
      c[7][0].w += reg_a[j % 2][1].w * reg_b[j % 2][0].w;
      c[7][1].x += reg_a[j % 2][1].w * reg_b[j % 2][1].x;
      c[7][1].y += reg_a[j % 2][1].w * reg_b[j % 2][1].y;
      c[7][1].z += reg_a[j % 2][1].w * reg_b[j % 2][1].z;
      c[7][1].w += reg_a[j % 2][1].w * reg_b[j % 2][1].w;
    }

    if (i < K) {
      *((float *)&smem_a[write_stage_idx][0] + sts_a_offset + 0 * TILE_Y + 0) =
          ldg_a_reg[0].x;
      *((float *)&smem_a[write_stage_idx][0] + sts_a_offset + 1 * TILE_Y + 0) =
          ldg_a_reg[0].y;
      *((float *)&smem_a[write_stage_idx][0] + sts_a_offset + 2 * TILE_Y + 0) =
          ldg_a_reg[0].z;
      *((float *)&smem_a[write_stage_idx][0] + sts_a_offset + 3 * TILE_Y + 0) =
          ldg_a_reg[0].w;
      *((float *)&smem_a[write_stage_idx][0] + sts_a_offset + 0 * TILE_Y + 64) =
          ldg_a_reg[1].x;
      *((float *)&smem_a[write_stage_idx][0] + sts_a_offset + 1 * TILE_Y + 64) =
          ldg_a_reg[1].y;
      *((float *)&smem_a[write_stage_idx][0] + sts_a_offset + 2 * TILE_Y + 64) =
          ldg_a_reg[1].z;
      *((float *)&smem_a[write_stage_idx][0] + sts_a_offset + 3 * TILE_Y + 64) =
          ldg_a_reg[1].w;

      smem_b[write_stage_idx][sts_b_offset + 0] = ldg_b_reg[0];
      smem_b[write_stage_idx][sts_b_offset + 8 * TILE_X_4] = ldg_b_reg[1];
      __syncthreads();
      write_stage_idx ^= 1;
    }

    reg_a[0][0] = smem_a[load_stage_idx ^ 1][0 + ty];
    reg_a[0][1] = smem_a[load_stage_idx ^ 1][16 + ty];
    reg_b[0][0] = smem_b[load_stage_idx ^ 1][0 + tx];
    reg_b[0][1] = smem_b[load_stage_idx ^ 1][16 + tx];

    c[0][0].x += reg_a[1][0].x * reg_b[1][0].x;
    c[0][0].y += reg_a[1][0].x * reg_b[1][0].y;
    c[0][0].z += reg_a[1][0].x * reg_b[1][0].z;
    c[0][0].w += reg_a[1][0].x * reg_b[1][0].w;
    c[0][1].x += reg_a[1][0].x * reg_b[1][1].x;
    c[0][1].y += reg_a[1][0].x * reg_b[1][1].y;
    c[0][1].z += reg_a[1][0].x * reg_b[1][1].z;
    c[0][1].w += reg_a[1][0].x * reg_b[1][1].w;
    c[1][0].x += reg_a[1][0].y * reg_b[1][0].x;
    c[1][0].y += reg_a[1][0].y * reg_b[1][0].y;
    c[1][0].z += reg_a[1][0].y * reg_b[1][0].z;
    c[1][0].w += reg_a[1][0].y * reg_b[1][0].w;
    c[1][1].x += reg_a[1][0].y * reg_b[1][1].x;
    c[1][1].y += reg_a[1][0].y * reg_b[1][1].y;
    c[1][1].z += reg_a[1][0].y * reg_b[1][1].z;
    c[1][1].w += reg_a[1][0].y * reg_b[1][1].w;
    c[2][0].x += reg_a[1][0].z * reg_b[1][0].x;
    c[2][0].y += reg_a[1][0].z * reg_b[1][0].y;
    c[2][0].z += reg_a[1][0].z * reg_b[1][0].z;
    c[2][0].w += reg_a[1][0].z * reg_b[1][0].w;
    c[2][1].x += reg_a[1][0].z * reg_b[1][1].x;
    c[2][1].y += reg_a[1][0].z * reg_b[1][1].y;
    c[2][1].z += reg_a[1][0].z * reg_b[1][1].z;
    c[2][1].w += reg_a[1][0].z * reg_b[1][1].w;
    c[3][0].x += reg_a[1][0].w * reg_b[1][0].x;
    c[3][0].y += reg_a[1][0].w * reg_b[1][0].y;
    c[3][0].z += reg_a[1][0].w * reg_b[1][0].z;
    c[3][0].w += reg_a[1][0].w * reg_b[1][0].w;
    c[3][1].x += reg_a[1][0].w * reg_b[1][1].x;
    c[3][1].y += reg_a[1][0].w * reg_b[1][1].y;
    c[3][1].z += reg_a[1][0].w * reg_b[1][1].z;
    c[3][1].w += reg_a[1][0].w * reg_b[1][1].w;
    c[4][0].x += reg_a[1][1].x * reg_b[1][0].x;
    c[4][0].y += reg_a[1][1].x * reg_b[1][0].y;
    c[4][0].z += reg_a[1][1].x * reg_b[1][0].z;
    c[4][0].w += reg_a[1][1].x * reg_b[1][0].w;
    c[4][1].x += reg_a[1][1].x * reg_b[1][1].x;
    c[4][1].y += reg_a[1][1].x * reg_b[1][1].y;
    c[4][1].z += reg_a[1][1].x * reg_b[1][1].z;
    c[4][1].w += reg_a[1][1].x * reg_b[1][1].w;
    c[5][0].x += reg_a[1][1].y * reg_b[1][0].x;
    c[5][0].y += reg_a[1][1].y * reg_b[1][0].y;
    c[5][0].z += reg_a[1][1].y * reg_b[1][0].z;
    c[5][0].w += reg_a[1][1].y * reg_b[1][0].w;
    c[5][1].x += reg_a[1][1].y * reg_b[1][1].x;
    c[5][1].y += reg_a[1][1].y * reg_b[1][1].y;
    c[5][1].z += reg_a[1][1].y * reg_b[1][1].z;
    c[5][1].w += reg_a[1][1].y * reg_b[1][1].w;
    c[6][0].x += reg_a[1][1].z * reg_b[1][0].x;
    c[6][0].y += reg_a[1][1].z * reg_b[1][0].y;
    c[6][0].z += reg_a[1][1].z * reg_b[1][0].z;
    c[6][0].w += reg_a[1][1].z * reg_b[1][0].w;
    c[6][1].x += reg_a[1][1].z * reg_b[1][1].x;
    c[6][1].y += reg_a[1][1].z * reg_b[1][1].y;
    c[6][1].z += reg_a[1][1].z * reg_b[1][1].z;
    c[6][1].w += reg_a[1][1].z * reg_b[1][1].w;
    c[7][0].x += reg_a[1][1].w * reg_b[1][0].x;
    c[7][0].y += reg_a[1][1].w * reg_b[1][0].y;
    c[7][0].z += reg_a[1][1].w * reg_b[1][0].z;
    c[7][0].w += reg_a[1][1].w * reg_b[1][0].w;
    c[7][1].x += reg_a[1][1].w * reg_b[1][1].x;
    c[7][1].y += reg_a[1][1].w * reg_b[1][1].y;
    c[7][1].z += reg_a[1][1].w * reg_b[1][1].z;
    c[7][1].w += reg_a[1][1].w * reg_b[1][1].w;

  } while (i < K);

#pragma unroll
  for (int wm = 0; wm < WPTM; wm++) {
#pragma unroll
    for (int wn = 0; wn < WPTN_4; wn++) {
      c[wm][wn].x *= alpha;
      c[wm][wn].y *= alpha;
      c[wm][wn].z *= alpha;
      c[wm][wn].w *= alpha;
    }
  }

#pragma unroll
  for (int wm = 0; wm < 4; wm++) {
#pragma unroll
    for (int wn = 0; wn < WPTN_4; wn++) {
      if (((blockIdx.y * TILE_Y + ty * 4 + wm) < M) &&
          ((blockIdx.x * TILE_X + wn * 64 + tx * 4) < N)) {
        if (beta != 0) {
          float4 vec4c = *(pC + ((ty * 4 + wm) * N / 4 + wn * 16 + tx));
          vec4c.x = vec4c.x * beta + c[wm][wn].x;
          vec4c.y = vec4c.y * beta + c[wm][wn].y;
          vec4c.z = vec4c.z * beta + c[wm][wn].z;
          vec4c.w = vec4c.w * beta + c[wm][wn].w;
          *(pC + (ty * 4 + wm) * N / 4 + wn * 16 + tx) = vec4c;
        } else {
          *(pC + (ty * 4 + wm) * N / 4 + wn * 16 + tx) = c[wm][wn];
        }
      }
    }
  }

#pragma unroll
  for (int wm = 0; wm < 4; wm++) {
#pragma unroll
    for (int wn = 0; wn < WPTN_4; wn++) {
      if (((blockIdx.y * TILE_Y + 64 + ty * 4 + wm) < M) &&
          ((blockIdx.x * TILE_X + wn * 64 + tx * 4) < N)) {
        if (beta != 0) {
          float4 vec4c = *(pC + ((64 + ty * 4 + wm) * N / 4 + wn * 16 + tx));
          vec4c.x = vec4c.x * beta + c[wm + 4][wn].x;
          vec4c.y = vec4c.y * beta + c[wm + 4][wn].y;
          vec4c.z = vec4c.z * beta + c[wm + 4][wn].z;
          vec4c.w = vec4c.w * beta + c[wm + 4][wn].w;
          *(pC + (64 + ty * 4 + wm) * N / 4 + wn * 16 + tx) = vec4c;
        } else {
          *(pC + (64 + ty * 4 + wm) * N / 4 + wn * 16 + tx) = c[wm + 4][wn];
        }
      }
    }
  }
}

int main(void) {
  int m = 2048;
  int n = 2048;
  int k = 1024;
  DATATYPE *a, *b;
  hipError_t status = hipHostMalloc(&a, sizeof(DATATYPE) * m * k, hipHostMallocDefault);
  if (status != hipSuccess) {
    printf("分配paged内存失败");
  }
  status = hipHostMalloc(&b, sizeof(DATATYPE) * k * n, hipHostMallocDefault);
  if (status != hipSuccess) {
    printf("分配paged内存失败");
  }
  init(a, m * k);
  init(b, k * n);

  ACCU_DATATYPE *c;
  hipHostMalloc(&c, sizeof(ACCU_DATATYPE) * m * n, hipHostMallocDefault);
  memset(c, 0, sizeof(ACCU_DATATYPE) * m * n);

  float *c_cpu_fp32 = (float *)malloc(sizeof(float) * m * n);
  memset(c_cpu_fp32, 0, sizeof(float) * m * n);

  DATATYPE *dev_a, *dev_b;
  float4 *dev_c_float4;
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // allocate the memory on the GPU
  double time1 = (double)clock() / CLOCKS_PER_SEC;
  using std::chrono::system_clock;
  system_clock::time_point today = system_clock::now();

  hipMalloc((void **)&dev_a, m * k * sizeof(DATATYPE));
  hipMalloc((void **)&dev_b, k * n * sizeof(DATATYPE));
  hipMalloc((void **)&dev_c_float4, m * n * sizeof(ACCU_DATATYPE));

  hipMemcpy(dev_a, a, m * k * sizeof(DATATYPE), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, k * n * sizeof(DATATYPE), hipMemcpyHostToDevice);

  uint3 grid = {m / (128), n / (128), 1};
  uint3 block = {256, 1, 1};

  for (int i = 0; i < WARMUP; i++) {
    gemm_kernel_NN<<<grid, block>>>(dev_a, dev_b, dev_c_float4, 1, 0, m, n, k);
  }

  hipEvent_t beg, end;
  hipEventCreate(&beg);
  hipEventCreate(&end);
  hipEventRecord(beg);

  for (int i = 0; i < REPEATE; i++) {
    gemm_kernel_NN<<<grid, block>>>(dev_a, dev_b, dev_c_float4, 1, 0, m, n, k);
  }

  hipEventRecord(end);
  hipEventSynchronize(end);
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, beg, end);
  printf("gpu gemm compute time: %f\n", elapsed_time);

  hipMemcpy(c, dev_c_float4, m * n * sizeof(ACCU_DATATYPE),
             hipMemcpyDeviceToHost);

  double time2 = (double)clock() / CLOCKS_PER_SEC;
  system_clock::time_point now = system_clock::now();
  auto ts = std::chrono::duration_cast<std::chrono::microseconds>(now - today);
  std::cout << "gpu total time:" << ts.count() / 1000.0 << "ms" << std::endl;
  printf("gpu total time:%lf\n", double(time2 - time1) * 1000);

  time1 = (double)clock() / CLOCKS_PER_SEC;
  today = system_clock::now();

  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      double sum = 0.f;
      for (int ii = 0; ii < k; ii++) {
#if DATATYPE_BYTE == 4
        sum += a[i * k + ii] * b[ii * n + j];
#else
        sum += __half2float(a[i * k + ii]) * __half2float(b[ii * n + j]);
#endif
      }
      c_cpu_fp32[i * n + j] = sum;
    }
  }

  time2 = (double)clock() / CLOCKS_PER_SEC;
  now = system_clock::now();
  ts = std::chrono::duration_cast<std::chrono::microseconds>(now - today);
  std::cout << "cpu time:" << ts.count() / 1000.0 << "ms" << std::endl;
  printf("cpu time:%lf\n", double(time2 - time1) * 1000);

  double max_diff = -1.;
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
#if ACCU_DATATYPE_BYTE == 4
      double c_gpu_fp32 = c[i * n + j];
#else
      double c_gpu_fp32 = __half2float(c[i * n + j]);
#endif
      if (std::abs(c_cpu_fp32[i * n + j] - c_gpu_fp32) > max_diff) {
        max_diff = std::abs(c_cpu_fp32[i * n + j] - c_gpu_fp32);
      }
    }
  }

  printf("max_diff: %f\n", max_diff);

  hipDeviceReset();
  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);
  free(c_cpu_fp32);
  return 0;
}