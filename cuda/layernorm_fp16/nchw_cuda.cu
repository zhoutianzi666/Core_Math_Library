#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>

template <typename T>
using kvp = hipcub::KeyValuePair<T, T>;

template <int VPT>
struct BytesToType;

template <>
struct BytesToType<2> {
  using type = uint16_t;
};
template <>
struct BytesToType<4> {
  using type = uint32_t;
};
template <>
struct BytesToType<8> {
  using type = uint64_t;
};
template <>
struct BytesToType<16> {
  using type = float4;
};

template <int Bytes>
__device__ inline void copy(const void* local, void* data) {
  using T = typename BytesToType<Bytes>::type;

  const T* in = static_cast<const T*>(local);
  T* out = static_cast<T*>(data);
  *out = *in;
}

template <typename T>
struct mySum {
  __host__ __device__ __forceinline__ kvp<T> operator()(kvp<T> const& a,
                                                        kvp<T> const& b) const {
    return kvp<T>(a.key + b.key, a.value + b.value);
  }
};

/*

template <typename T, typename OP_T, int32_t TPB>
__global__ void LayerNormSmallKernel(int32_t const nHiddenDimension,
                                     T const* input,
                                     T const* gamma,
                                     T const* beta,
                                     T* output,
                                     float const epsilon) {
  int32_t const index = blockIdx.x * nHiddenDimension + threadIdx.x;
  T const denominator = T(1.f) ;/// T(nHiddenDimension);
  OP_T val = 0;
  kvp<OP_T> threadData(0, 0);

  if (threadIdx.x < nHiddenDimension) {
    val = input[index];
    OP_T tmp0 = 0;//input[index] * denominator;
    OP_T tmp1 = tmp0 * static_cast<OP_T>(input[index]);
    threadData = mySum<OP_T>()(threadData, kvp<OP_T>(tmp0, tmp1));
  }

  using WarpReduce = hipcub::WarpReduce<kvp<OP_T>, TPB>;
  __shared__ typename WarpReduce::TempStorage temp;
  __shared__ OP_T mu, rsigma;

  auto const sumKV = WarpReduce(temp).Reduce(threadData, mySum<OP_T>());
  if (threadIdx.x == 0) {
    mu = sumKV.key;
    printf("%f\n", mu);
    rsigma = rsqrt(sumKV.value - mu * mu + static_cast<OP_T>(epsilon));
  }
  __syncthreads();

  if (threadIdx.x < nHiddenDimension) {
    OP_T const g = gamma[threadIdx.x], b = beta[threadIdx.x];
    output[index] = (val - mu) * rsigma * g + b;
  }
}

*/

// template __global__ void LayerNormSmallKernel<float, float, 32>(int32_t const,
//                                                                 float const*,
//                                                                 float const*,
//                                                                 float const*,
//                                                                 float*,
//                                                                 float const);
// template __global__ void LayerNormSmallKernel<__half, float, 32>(int32_t const,
//                                                                  __half const*,
//                                                                  __half const*,
//                                                                  __half const*,
//                                                                  __half*,
//                                                                  float const);

template <typename T, typename OP_T, int32_t TPB, int32_t VPT>
__global__ void LayerNormMediumKernel(int32_t const nHiddenDimension,
                                      T const* input,
                                      T const* gamma,
                                      T const* beta,
                                      T* output,
                                      float const epsilon) {
  int32_t const index = blockIdx.x * nHiddenDimension + threadIdx.x * VPT;
  T localX[VPT], localGamma[VPT], localBeta[VPT];
  OP_T const denominator = OP_T(1) / OP_T(nHiddenDimension);
  kvp<OP_T> threadData(0, 0);

  copy<sizeof(T) * VPT>(&input[index], localX);
#pragma unroll
  for (int32_t it = 0; it < VPT; it++) {
    OP_T const tmp = static_cast<OP_T>(localX[it]) * denominator;
    threadData = mySum<OP_T>()(
        threadData, kvp<OP_T>(tmp, tmp * static_cast<OP_T>(localX[it])));
  }

  copy<sizeof(T) * VPT>(&beta[threadIdx.x * VPT], localBeta);
  copy<sizeof(T) * VPT>(&gamma[threadIdx.x * VPT], localGamma);

  using BlockReduce = hipcub::BlockReduce<kvp<OP_T>, TPB>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ OP_T mu, rsigma;

  auto const sumKV =
      BlockReduce(temp_storage).Reduce(threadData, mySum<OP_T>());
  if (threadIdx.x == 0) {
    mu = sumKV.key;
    rsigma = rsqrt(sumKV.value - mu * mu + static_cast<OP_T>(epsilon));
  }
  __syncthreads();

#pragma unroll
  for (int32_t it = 0; it < VPT; it++) {
    localX[it] = static_cast<OP_T>(localGamma[it]) *
                     (static_cast<OP_T>(localX[it]) - mu) * rsigma +
                 static_cast<OP_T>(localBeta[it]);
  }

  copy<sizeof(T) * VPT>(localX, &output[index]);
}

template __global__ void LayerNormMediumKernel<float, float, 64, 4>(
    int32_t const,
    float const*,
    float const*,
    float const*,
    float*,
    float const);
template __global__ void LayerNormMediumKernel<__half, float, 64, 4>(
    int32_t const,
    __half const*,
    __half const*,
    __half const*,
    __half*,
    float const);

template <typename T, typename OP_T, int32_t TPB>
__global__ void LayerNormLargeKernel(int32_t const nHiddenDimension,
                                     T const* input,
                                     T const* gamma,
                                     T const* beta,
                                     T* output,
                                     float const epsilon) {
  int32_t const offset = blockIdx.x * nHiddenDimension;
  OP_T const denominator = OP_T(1) / OP_T(nHiddenDimension);
  kvp<OP_T> threadData(0, 0);

  for (int32_t i = threadIdx.x; i < nHiddenDimension; i += TPB) {
    int32_t const index = offset + i;
    OP_T val = input[index];
    OP_T const tmp = val * denominator;
    threadData = mySum<OP_T>()(threadData, kvp<OP_T>(tmp, tmp * val));
    output[index] = val;
  }

  using BlockReduce = hipcub::BlockReduce<kvp<OP_T>, TPB>;
  __shared__ typename BlockReduce::TempStorage temp;
  __shared__ OP_T mu, rsigma;

  auto const sumKV = BlockReduce(temp).Reduce(threadData, mySum<OP_T>());

  if (threadIdx.x == 0) {
    mu = sumKV.key;
    rsigma = rsqrt(sumKV.value - mu * mu + static_cast<OP_T>(epsilon));
  }
  __syncthreads();

  for (int32_t i = threadIdx.x; i < nHiddenDimension; i += TPB) {
    int32_t const index = offset + i;
    output[index] = (static_cast<OP_T>(output[index]) - mu) * rsigma *
                        static_cast<OP_T>(gamma[i]) +
                    static_cast<OP_T>(beta[i]);
  }
}

template __global__ void LayerNormLargeKernel<float, float, 256>(int32_t const,
                                                                 float const*,
                                                                 float const*,
                                                                 float const*,
                                                                 float*,
                                                                 float const);
template __global__ void LayerNormLargeKernel<__half, float, 256>(int32_t const,
                                                                  __half const*,
                                                                  __half const*,
                                                                  __half const*,
                                                                  __half*,
                                                                  float const);

template <typename T>
int32_t layernorm_gpu(int32_t const gridSize,
                         int32_t const nHiddenDimension,
                         T const* input,
                         T const* gamma,
                         T const* beta,
                         T* output,
                         float const epsilon) {
  constexpr int32_t VPT = 16 / sizeof(T);
  if (nHiddenDimension <= 32) {
    constexpr int32_t TPB = 32;
    // (LayerNormSmallKernel<T, float, TPB>)<<<gridSize, TPB, 0>>>(
    //     nHiddenDimension, input, gamma, beta, output, epsilon);
  } else if (nHiddenDimension == 320) {
    constexpr int32_t TPB = 320 / VPT;
    (LayerNormMediumKernel<T, float, TPB, VPT>)<<<gridSize, TPB, 0>>>(
        nHiddenDimension, input, gamma, beta, output, epsilon);
  } else if (nHiddenDimension == 640) {
    constexpr int32_t TPB = 640 / VPT;
    (LayerNormMediumKernel<T, float, TPB, VPT>)<<<gridSize, TPB, 0>>>(
        nHiddenDimension, input, gamma, beta, output, epsilon);
  } else if (nHiddenDimension == 768) {
    constexpr int32_t TPB = 768 / VPT;
    (LayerNormMediumKernel<T, float, TPB, VPT>)<<<gridSize, TPB, 0>>>(
        nHiddenDimension, input, gamma, beta, output, epsilon);
  } else if (nHiddenDimension == 1280) {
    constexpr int32_t TPB = 1280 / VPT;
    (LayerNormMediumKernel<T, float, TPB, VPT>)<<<gridSize, TPB, 0>>>(
        nHiddenDimension, input, gamma, beta, output, epsilon);
  } else {
    constexpr int32_t TPB = 256;
    (LayerNormLargeKernel<T, float, TPB>)<<<gridSize, TPB, 0>>>(
        nHiddenDimension, input, gamma, beta, output, epsilon);
  }
  // PLUGIN_CHECK_CUDA(hipPeekAtLastError());
  return 0;
}

template int layernorm_gpu<half>(int const,
                                    int const,
                                    half const*,
                                    half const*,
                                    half const*,
                                    half*,
                                    float const);
