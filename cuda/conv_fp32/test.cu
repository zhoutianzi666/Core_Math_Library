#include "hip/hip_runtime.h"

#include <stdio.h>

#include <chrono>
#include <ctime>
#include <iostream>
#include <ratio>

#include "hipblas.h"
#include <cudnn_v8.h>
#include "utility.h"

#define WARMUP 10
#define REPEATE 10

using DATATYPE = float;
using C_DATATYPE = float;

int main(void) {
  int batch = 1;
  int ic = 32;
  int ih = 112;
  int iw = 112;
  int pad_h = 1;
  int pad_w = 1;
  int oc = 32;
  int kh = 3;
  int kw = 3;
  int stride_h = 1;
  int stride_w = 1;

  // Here is consistent with
  int oh = (ih + pad_h * 2 - kh) / stride_h + 1;
  int ow = (iw + pad_w * 2 - kw) / stride_w + 1;

  // Note input and weight is in CPU place
  DATATYPE *input, *weight;
  int input_size = batch * ic * ih * iw;
  int weight_size = oc * ic * kh * kw;

  hipError_t status = hipHostMalloc(&input, sizeof(DATATYPE) * input_size);
  status = hipHostMalloc(&weight, sizeof(DATATYPE) * weight_size);
  init(input, input_size);
  init(weight, weight_size);

  // out is used to store the result form dev_out
  C_DATATYPE *out_from_dev;
  int out_size = batch * oc * oh * ow;
  hipHostMalloc(&out_from_dev, sizeof(C_DATATYPE) * out_size);
  memset(out_from_dev, 0, sizeof(C_DATATYPE) * out_size);

  // dev_input is from weight
  // dev_weight is from weight
  DATATYPE *dev_input, *dev_weight;
  C_DATATYPE *dev_out;

  hipdnnHandle_t handle_cudnn;
  hipdnnCreate(&handle_cudnn);
  hipdnnTensorDescriptor_t input_descriptor;
  auto cudnn_state = hipdnnCreateTensorDescriptor(&input_descriptor);
  hipdnnSetTensor4dDescriptor(input_descriptor,
                             HIPDNN_TENSOR_NHWC,
                             HIPDNN_DATA_FLOAT, batch, ic, ih, iw);
  hipdnnTensorDescriptor_t output_descriptor;
  cudnn_state=hipdnnCreateTensorDescriptor(&output_descriptor);
  cudnn_state=hipdnnSetTensor4dDescriptor(output_descriptor,
                            HIPDNN_TENSOR_NHWC,
                            HIPDNN_DATA_FLOAT, batch, oc, oh, ow);
  hipdnnFilterDescriptor_t kernel_descriptor;
  cudnn_state=hipdnnCreateFilterDescriptor(&kernel_descriptor);
  cudnn_state=hipdnnSetFilter4dDescriptor(kernel_descriptor,
                              HIPDNN_DATA_FLOAT,
                              HIPDNN_TENSOR_NHWC,
                              oc, ic, kh, kw);

  hipdnnConvolutionDescriptor_t conv_descriptor;
  cudnn_state= hipdnnCreateConvolutionDescriptor(&conv_descriptor);
  cudnn_state = hipdnnSetConvolution2dDescriptor(conv_descriptor,
                                  1, 1, // zero-padding
                                  1, 1, // stride
                                  1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);


int returnedAlgoCount;
hipdnnConvolutionFwdAlgoPerf_t     perfResults[100];
hipdnnFindConvolutionForwardAlgorithm(handle_cudnn, input_descriptor, kernel_descriptor, conv_descriptor,
  output_descriptor, 100, &returnedAlgoCount, perfResults);


  printf("%d\n", returnedAlgoCount);
size_t workspace_size = 0;
cudnn_state = hipdnnGetConvolutionForwardWorkspaceSize(handle_cudnn,
                                        input_descriptor,
                                        kernel_descriptor,
                                        conv_descriptor,
                                        output_descriptor,
                                        perfResults[0].algo,
                                        &workspace_size);
printf("%d\n", workspace_size);
hipdnnSetConvolutionMathType(conv_descriptor ,HIPDNN_TENSOR_OP_MATH);
void * workspace = nullptr;
hipMalloc(&workspace, workspace_size);

  // allocate the memory on the GPU
  double time1 = (double)clock() / CLOCKS_PER_SEC;
  using std::chrono::system_clock;
  system_clock::time_point today = system_clock::now();

  hipMalloc((void **)&dev_input, input_size * sizeof(DATATYPE));
  hipMalloc((void **)&dev_weight, weight_size * sizeof(DATATYPE));
  hipMalloc((void **)&dev_out, out_size * sizeof(C_DATATYPE));

  hipMemcpy(dev_input, input, input_size * sizeof(DATATYPE),
             hipMemcpyHostToDevice);
  hipMemcpy(dev_weight, weight, weight_size * sizeof(DATATYPE),
             hipMemcpyHostToDevice);
  hipStream_t stream = nullptr;
  for (int i = 0; i < WARMUP; i++) {

  //   const float alpha = 1.0f;
  //   const float beta = 0.0f;
  //   cudnn_state = hipdnnConvolutionForward(
  // handle_cudnn,
  // &alpha,
  // input_descriptor,
  // dev_input,
  // kernel_descriptor,
  // dev_weight,
  // conv_descriptor,
  // perfResults[0].algo,
  // workspace,
  // workspace_size,
  // &beta,
  // output_descriptor,
  // dev_out);
  cutlass_nhwc_conv(dev_input, dev_weight, dev_out, batch, ic, ih, iw, kh, kw,
    oc, pad_h, pad_w, stride_h, stride_w, oh, ow, stream);
  }

  hipEvent_t beg, end;
  hipEventCreate(&beg);
  hipEventCreate(&end);
  hipEventRecord(beg);

  for (int i = 0; i < REPEATE; i++) {

    // const float alpha = 1.0f;
    // const float beta = 0.0f;
    // hipdnnConvolutionForward(
    // handle_cudnn,
    // &alpha,
    // input_descriptor,
    // dev_input,
    // kernel_descriptor,
    // dev_weight,
    // conv_descriptor,
    // perfResults[0].algo,
    // workspace,
    // workspace_size,
    // &beta,
    // output_descriptor,
    // dev_out);
    cutlass_nhwc_conv(dev_input, dev_weight, dev_out, batch, ic, ih, iw, kh, kw,
      oc, pad_h, pad_w, stride_h, stride_w, oh, ow, stream);
  }

  hipEventRecord(end);
  hipEventSynchronize(end);
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, beg, end);
  printf("gpu conv compute time: %f\n", elapsed_time);
  double Gflops =
      REPEATE * ((float)out_size * ic * kh * kw * 2 / 1000000) / elapsed_time;
  printf("Gflops: %5.2f \n", Gflops);

  hipMemcpy(out_from_dev, dev_out, out_size * sizeof(C_DATATYPE),
             hipMemcpyDeviceToHost);

  double time2 = (double)clock() / CLOCKS_PER_SEC;
  system_clock::time_point now = system_clock::now();
  auto ts = std::chrono::duration_cast<std::chrono::microseconds>(now - today);
  std::cout << "gpu total time:" << ts.count() / 1000.0 << "ms" << std::endl;
  printf("gpu total time:%lf\n", double(time2 - time1) * 1000);

  time1 = (double)clock() / CLOCKS_PER_SEC;
  today = system_clock::now();

  // results calculated in cpu is always fp32
  float *out_cpu_fp32 = (float *)malloc(sizeof(float) * out_size);
  memset(out_cpu_fp32, 0, sizeof(float) * out_size);

  naive_conv_cpu(input, weight, out_cpu_fp32, batch, ic, ih, iw, kh, kw, oc,
                 pad_h, pad_w, stride_h, stride_w);

  time2 = (double)clock() / CLOCKS_PER_SEC;
  now = system_clock::now();
  ts = std::chrono::duration_cast<std::chrono::microseconds>(now - today);
  std::cout << "cpu time:" << ts.count() / 1000.0 << "ms" << std::endl;
  printf("cpu time:%lf\n", double(time2 - time1) * 1000);

  printf("max_diff: %f\n", diff(out_from_dev, out_cpu_fp32, out_size));

  hipDeviceReset();
  hipHostFree(input);
  hipHostFree(weight);
  hipHostFree(out_from_dev);
  free(out_cpu_fp32);
  return 0;
}
