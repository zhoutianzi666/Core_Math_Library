#pragma once
#include <stdio.h>

#include <iostream>

#include "hipblas.h"
#include "cutlass/gemm/device/gemm.h"
#include "utility.h"

#include <algorithm>

#include "cutlass/conv/device/implicit_gemm_convolution.h"
#include "cutlass/conv/kernel/default_conv2d_fprop.h"
#include "cutlass/cutlass.h"
#include "cutlass/epilogue/thread/linear_combination_leaky_relu.h"

static cutlass::conv::IteratorAlgorithm const IteratorAlgorithm =
    cutlass::conv::IteratorAlgorithm::kOptimized;

using EpilogueOp = cutlass::epilogue::thread::LinearCombinationLeakyRelu<
    cutlass::half_t,  // Data type of output matrix.
    8,
    float,   // Data type of accumulator
    float>;  // Data type for alpha/beta in linear combination

using Conv2dFpropKernel = typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t, cutlass::layout::TensorNHWC, cutlass::half_t,
    cutlass::layout::TensorNHWC, cutlass::half_t, cutlass::layout::TensorNHWC,
    float, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<32, 32, 64>,
    cutlass::gemm::GemmShape<16, 8, 8>, EpilogueOp,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, 2,
    cutlass::arch::OpMultiplyAdd, IteratorAlgorithm,
    cutlass::conv::StrideSupport::kUnity, 1, 1>::Kernel;

using ImplicitGemm =
    cutlass::conv::device::ImplicitGemmConvolution<Conv2dFpropKernel>;

void cutlass_nhwc_conv_bias_leaky_relu(ConvAllParams params) {

  int batch = params.batch;
  int ih = params.ih;
  int iw = params.iw;
  int ic = params.ic;
  int oc = params.oc;
  int kh = params.kh;
  int kw = params.kw;
  int pad_h0 = params.pad_h0;
  int pad_h1 = params.pad_h1;
  int pad_w0 = params.pad_w0;
  int pad_w1 = params.pad_w1;
  int stride_h = params.stride_h;
  int stride_w = params.stride_w;
  int dilation_h = params.dilation_h;
  int dilation_w = params.dilation_w;

  int oh = params.oh;
  int ow = params.ow;
  auto input = params.input;
  auto weight = params.weight;
  auto bias = params.bias;
  auto output = params.output;

  int groups = 1;

  cutlass::conv::Mode mode = cutlass::conv::Mode::kCrossCorrelation;

  cutlass::conv::Conv2dProblemSize problem_size(
      {batch, ih, iw, ic}, {oc, kh, kw, ic}, {pad_h0, 0, pad_w0, 0},
      {stride_h, stride_w}, {dilation_h, dilation_w}, {batch, oh, ow, oc}, mode,
      1, groups);

  typename ImplicitGemm::Arguments arguments{
      problem_size,
      {(cutlass::half_t *)input, {ic, ic * iw, ic * iw * ih}},
      {(cutlass::half_t *)weight, {ic, ic * kw, ic * kw * kh}},
      {(cutlass::half_t *)bias, {0, 0, 0}},
      {(cutlass::half_t *)output, {oc, oc * ow, oc * ow * oh}},
      {1.f, 1.f, 0.5f},
      cutlass::conv::SplitKMode::kParallel};

  ImplicitGemm implicit_gemm_op;
  size_t bytes = implicit_gemm_op.get_workspace_size(arguments);
  void *workspace;
  hipMalloc((void **)&workspace, bytes);

  cutlass::Status status = implicit_gemm_op.can_implement(arguments);
  CUTLASS_CHECK(status);
  status = implicit_gemm_op.initialize(arguments, workspace);
  CUTLASS_CHECK(status);
  status = implicit_gemm_op();
  CUTLASS_CHECK(status);
  hipFree(workspace);
}
