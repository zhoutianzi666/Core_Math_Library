#pragma once
#include <stdio.h>

#include <iostream>

#include "hipblas.h"
#include "cutlass/gemm/device/gemm.h"
#include "utility.h"

using DATATYPE = half;

#include <algorithm>

#include "cutlass/conv/device/implicit_gemm_convolution.h"
#include "cutlass/conv/kernel/default_conv2d_fprop.h"
#include "cutlass/cutlass.h"
#include "cutlass/epilogue/thread/linear_combination_silu.h"

static cutlass::conv::IteratorAlgorithm const IteratorAlgorithm =
    cutlass::conv::IteratorAlgorithm::kFewChannels;

using EpilogueOp = cutlass::epilogue::thread::LinearCombinationSilu<
    cutlass::half_t,  // Data type of output matrix.
    1,
    float,   // Data type of accumulator
    float>;  // Data type for alpha/beta in linear combination

using Conv2dFpropKernel = typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t, cutlass::layout::TensorNHWC, cutlass::half_t,
    cutlass::layout::TensorNHWC, cutlass::half_t, cutlass::layout::TensorNHWC,
    float, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<16, 32, 16>, cutlass::gemm::GemmShape<16, 32, 16>,
    cutlass::gemm::GemmShape<16, 8, 8>, EpilogueOp,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, 2,
    cutlass::arch::OpMultiplyAdd, IteratorAlgorithm,
    cutlass::conv::StrideSupport::kUnity, 8, 8>::Kernel;

using ImplicitGemm =
    cutlass::conv::device::ImplicitGemmConvolution<Conv2dFpropKernel>;

void cutlass_nhwc_conv_bias_swish(const half *input, const half *weight,
                                  const half *bias, half *output, int batch,
                                  int ic, int ih, int iw, int kh, int kw,
                                  int oc, int pad_h, int pad_w, int stride_h,
                                  int stride_w, int oh, int ow) {
  auto check = [](cutlass::Status status) {
    if (status != cutlass::Status::kSuccess) {
      printf("不能实施\n");
    }
  };

  cutlass::conv::Mode mode = cutlass::conv::Mode::kCrossCorrelation;

  cutlass::conv::Conv2dProblemSize problem_size(
      {batch, ih, iw, ic}, {oc, kh, kw, ic}, {pad_h, pad_w, pad_h, pad_w},
      {stride_h, stride_w}, {1, 1}, {batch, oh, ow, oc}, mode, 1);

  typename ImplicitGemm::Arguments arguments{
      problem_size,
      {(cutlass::half_t *)input, {ic, ic * iw, ic * iw * ih}},
      {(cutlass::half_t *)weight, {ic, ic * kw, ic * kw * kh}},
      {(cutlass::half_t *)bias, {0, 0, 0}},
      {(cutlass::half_t *)output, {oc, oc * ow, oc * ow * oh}},
      {1.f, 1.f},
      cutlass::conv::SplitKMode::kParallel};

  ImplicitGemm implicit_gemm_op;
  size_t bytes = implicit_gemm_op.get_workspace_size(arguments);
  void *workspace;
  hipMalloc((void **)&workspace, bytes);

  cutlass::Status status = implicit_gemm_op.can_implement(arguments);
  check(status);
  status = implicit_gemm_op.initialize(arguments, workspace);
  check(status);
  status = implicit_gemm_op();
  check(status);
}
