#include "hip/hip_runtime.h"
#include <stdio.h>

#include "utility.h"
#include <iostream>
#include <vector>
#include <functional>

#include "cutlass/cutlass.h"

void init(half *a, int size) {
  for (int i = 0; i < size; i++) {
    a[i] = __float2half((rand() % 9999) / 10000.0 - 0.5);
  }
}
void init(float *a, int size) {
  for (int i = 0; i < size; i++) {
    a[i] = (rand() % 9999) / 10000.0 - 0.5;
  }
}

float diff(const half *c, const float *c_baseline, int n) {
  float max_diff = -1.;
  for (int i = 0; i < n; i++) {
    float c_value = __half2float(c[i]);
    if (std::abs(c_baseline[i] - c_value) > max_diff) {
      max_diff = std::abs(c_baseline[i] - c_value);
    }
  }
  return max_diff;
}

float diff(const float *c, const float *c_baseline, int n) {
  float max_diff = -1.;
  for (int i = 0; i < n; i++) {
    float c_value = c[i];
    if (std::abs(c_baseline[i] - c_value) > max_diff) {
      max_diff = std::abs(c_baseline[i] - c_value);
    }
  }
  return max_diff;
}

int nchw(struct logical_struct shape, struct logical_struct index) {
  return index.n * shape.c * shape.h * shape.w + index.c * shape.h * shape.w +
         index.h * shape.w + index.w;
}

int nhwc(struct logical_struct shape, struct logical_struct index) {
  return index.n * shape.h * shape.w * shape.c + index.h * shape.w * shape.c +
         index.w * shape.c + index.c;
}

void naive_conv_cpu(ConvAllParams params) {
  int batch = params.batch;
  int ic = params.ic;
  int ih = params.ih;
  int iw = params.iw;
  int pad_h0 = params.pad_h0;
  int pad_h1 = params.pad_h1;
  int pad_w0 = params.pad_w0;
  int pad_w1 = params.pad_w1;
  int oc = params.oc;
  int groups = params.groups;
  int kc = ic / groups;
  int kh = params.kh;
  int kw = params.kw;
  int stride_h = params.stride_h;
  int stride_w = params.stride_w;
  int dilation_h = params.dilation_h;
  int dilation_w = params.dilation_w;
  int oh = params.oh;
  int ow = params.ow;

  auto input = params.input;
  auto weight = params.weight;
  auto residual = params.residual;
  auto bias = params.bias;
  auto output = params.output_cpu_fp32;

  struct logical_struct input_shape {
    batch, ic, ih, iw
  };
  struct logical_struct output_shape {
    batch, oc, oh, ow
  };
  struct logical_struct weight_shape {
    oc, kc, kh, kw
  };

  for (int bs_i = 0; bs_i < batch; bs_i++) {
    for (int oc_i = 0; oc_i < oc; oc_i++) {
      for (int oh_i = 0; oh_i < oh; oh_i++) {
        for (int ow_i = 0; ow_i < ow; ow_i++) {
          struct logical_struct output_index {
            bs_i, oc_i, oh_i, ow_i
          };
          float *out_ptr = output + nhwc(output_shape, output_index);
          float sum = 0.f;

          for (int kh_i = 0; kh_i < kh; kh_i++) {
            for (int kw_i = 0; kw_i < kw; kw_i++) {
              int ih_i = oh_i * stride_h - pad_h0 + kh_i * dilation_h;
              int iw_i = ow_i * stride_w - pad_w0 + kw_i * dilation_w;
              if (ih_i < 0 || ih_i >= ih) continue;
              if (iw_i < 0 || iw_i >= iw) continue;
              
              int groups_i = (oc_i / (oc / groups));
              int ic_start = groups_i * kc;
              int ic_end = (groups_i + 1) * kc;

              for (int ic_i = ic_start; ic_i < ic_end; ic_i++) {

                struct logical_struct input_index {
                  bs_i, ic_i, ih_i, iw_i
                };
                struct logical_struct weight_index {
                  oc_i, ic_i - ic_start, kh_i, kw_i
                };
                const half *in_ptr = input + nhwc(input_shape, input_index);
                const half *weight_ptr =
                    weight + nhwc(weight_shape, weight_index);
                sum += __half2float(*in_ptr) * __half2float(*weight_ptr);
              }
            }
          }

          // bias
          sum += __half2float(*(bias + oc_i));
          float x = sum;
          switch (params.act_type) {
            case IDENTITY:
              *out_ptr = sum ;
              break;
            case SIGMOID:
              *out_ptr = 1/ (1 + std::exp(-x)) ;
              break;
            case RELU:
              *out_ptr = sum > 0 ? sum : 0.f;
              break;
            case SILU:
              *out_ptr = (x) * (1.f / (1 + exp(-x)));
              break;
            case LEAKY_RELU:
              if (x > 0) *out_ptr = x;
              else {
                *out_ptr = x * 0.5 ;
              }
              break;
            case CONV2D_BIAS_ADD_RELU:
              x += __half2float(*(residual + nhwc(output_shape, output_index)));
              *out_ptr = x > 0 ? x : 0.f;
            default:
              break;
          }
        }
      }
    }
  }
}



int ProfileToGetBestConfig(
    const std::vector<std::function<cutlass::Status(ConvAllParams)>> &all_func,
    const ConvAllParams &params) {

  constexpr int WARMUP = 10;
  constexpr int REPEAT = 100;
  float min_time = 100000.f;
  int min_time_index = -1;
  for (int i = 0; i < all_func.size(); i++) {
    cutlass::Status status;
    auto func = all_func[i];
    // When func has large diff, we will make it nullptr.
    if (!func) continue;

    for (int ii = 0; ii < WARMUP; ii++) {
      status = func(params);
    }

    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);
    hipEventRecord(beg);
    for (int ii = 0; ii < REPEAT; ii++) {
      status = func(params);
    }

    hipEventRecord(end);
    hipEventSynchronize(end);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, beg, end);
    if (elapsed_time < min_time && status == cutlass::Status::kSuccess) {
      min_time = elapsed_time;
      min_time_index = i;
      // debug code
      std::cout << "tactic " << i << "cost_time: " << elapsed_time << "ms." << std::endl;
    }
  }

  if (min_time_index < 0) {
    assert(0);
  }
  return min_time_index;
}
