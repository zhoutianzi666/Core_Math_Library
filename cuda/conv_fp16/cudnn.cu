#pragma once
#include <stdio.h>
#include <cudnn_v8.h>
#include <string>
#include <iostream>

#include "hipblas.h"
#include "cutlass/gemm/device/gemm.h"
#include "utility.h"

using C_DATATYPE = half;

using DATATYPE = half;
#include <algorithm>
#define warp_M 16
#define warp_N 8
#define warp_K 8
#define WARP_SIZE 32
using DATATYPE = half;

void CUDNN_CHECK(hipdnnStatus_t status) {
  if (status != HIPDNN_STATUS_SUCCESS) {
    printf("CUDNN 不能实施\n");
  }
}


void cudnn_nhwc_conv(ConvAllParams params) {

  auto handle_cudnn = params.handle_cudnn;
  int batch = params.batch;
  int ih = params.ih;
  int iw = params.iw;
  int ic = params.ic;
  int oc = params.oc;
  int kh = params.kh;
  int kw = params.kw;
  int pad_h0 = params.pad_h0;
  int pad_h1 = params.pad_h1;
  int pad_w0 = params.pad_w0;
  int pad_w1 = params.pad_w1;
  int stride_h = params.stride_h;
  int stride_w = params.stride_w;
  int dilation_h = params.dilation_h;
  int dilation_w = params.dilation_w;

  int oh = params.oh;
  int ow = params.ow;
  auto input = params.input;
  auto weight = params.weight;
  auto bias = params.bias;
  auto output = params.output;

  int groups = params.groups;
  int kc = ic / groups;

  int out_size = batch * oc * oh * ow;

  size_t cudnn_workspace_size = params.cudnn_workspace_size;
  void *cudnn_workspace = params.cudnn_workspace;

  hipdnnTensorDescriptor_t input_descriptor;
  auto cudnn_layout = HIPDNN_TENSOR_NHWC;
  CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_descriptor));
  CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_descriptor, cudnn_layout,
                                         HIPDNN_DATA_HALF, batch, ic, ih, iw));

  hipdnnTensorDescriptor_t output_descriptor;
  CUDNN_CHECK(hipdnnCreateTensorDescriptor(&output_descriptor));
  CUDNN_CHECK(hipdnnSetTensor4dDescriptor(output_descriptor, cudnn_layout,
                                         HIPDNN_DATA_HALF, batch, oc, oh, ow));
  hipdnnFilterDescriptor_t kernel_descriptor;
  CUDNN_CHECK(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  CUDNN_CHECK(hipdnnSetFilter4dDescriptor(kernel_descriptor, HIPDNN_DATA_HALF,
                                         cudnn_layout, oc, kc, kh, kw));

  hipdnnConvolutionDescriptor_t conv_descriptor;
  CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
  CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(
      conv_descriptor, pad_h0, pad_w0,  // 右边不需要告诉他,还是默认是对称的呢？
      stride_h, stride_w,               // stride
      dilation_h, dilation_w, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
  CUDNN_CHECK(hipdnnSetConvolutionGroupCount(conv_descriptor, groups));
  hipdnnSetConvolutionMathType(conv_descriptor, HIPDNN_TENSOR_OP_MATH);

//   std::cout << "这个我猜可能是逻辑判断来确定的最优算法！" << std::endl;
//   int returnedAlgoCount;
//  int requestedAlgoCount = 100;
//   hipdnnConvolutionFwdAlgoPerf_t perfResults[100];

//   CUDNN_CHECK(hipdnnFindConvolutionForwardAlgorithm(
//       handle_cudnn, input_descriptor, kernel_descriptor, conv_descriptor,
//       output_descriptor, requestedAlgoCount, &returnedAlgoCount, perfResults));


//   printf("\t返回的算法个数是：%d\n", returnedAlgoCount);

//   for (int i = 0; i < returnedAlgoCount; i++) {
//     std::cout << "\t perfResults[" << i<< "]:" << cudnnAlgoName(perfResults[i].algo) 
//             <<" " << perfResults[i].status
//             << std::endl;
//   }

// 我认为这个是不需要的，workspace我们预先给他吧！
//   CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(
//       handle_cudnn, input_descriptor, kernel_descriptor, conv_descriptor,
//       output_descriptor, perfResults[0].algo, &cudnn_workspace_size));


//   // ---------------------------this is also cuDNN-----------------------------------------------
//   std::cout << "这个我猜可能是真实的计算来确定的最优算法！" << std::endl;
//   CUDNN_CHECK(hipdnnFindConvolutionForwardAlgorithmEx(
//       handle_cudnn, input_descriptor, input, kernel_descriptor, weight,
//       conv_descriptor, output_descriptor, output, 100, &returnedAlgoCount,
//       perfResults, cudnn_workspace, cudnn_workspace_size));
//   for (int i = 0; i < returnedAlgoCount; i++) {
//   std::cout << "\tperfResults[" << i<< "]:" << cudnnAlgoName(perfResults[i].algo) 
//             <<" " << perfResults[i].status
//             << std::endl;
//   }
//   printf("\t返回的算法个数是：%d\n", returnedAlgoCount);
//   hipMemset(output, 0, sizeof(C_DATATYPE) * out_size);

  // 上面必须要清零！因为cuDNN要计算这个结果！

  // ------------------------cuDNN ends-----------------------------------------------------

    const float alpha = 1.0f;
    const float beta = 0.0f;
    CUDNN_CHECK(hipdnnConvolutionForward(
    handle_cudnn,
    &alpha, 
    input_descriptor, input,
    kernel_descriptor, weight,
    conv_descriptor, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM, cudnn_workspace, cudnn_workspace_size, &beta, 
    output_descriptor, output));
}




std::string cudnnAlgoName(hipdnnConvolutionFwdAlgo_t algo)
{
    switch (algo) {
    case HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM:
         return std::string("HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM");
    case HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM:
        return std::string("HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM");
    case HIPDNN_CONVOLUTION_FWD_ALGO_GEMM:
        return std::string("HIPDNN_CONVOLUTION_FWD_ALGO_GEMM");
    case HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT:
        return std::string("HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT");
        
    default:
        return std::string("");
    }
    return std::string("");
}

