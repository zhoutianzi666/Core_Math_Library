#include "hip/hip_runtime.h"
#include <stdio.h>

#include <chrono>
#include <ctime>
#include <iostream>
#include <ratio>

#include "utility.h"

#define WARMUP 10
#define REPEATE 10

using DATATYPE = half;

int main(void) {
  int batch = 1;
  int ic = 32;
  int ih = 1;
  int iw = 32;
  // Note input is in CPU place
  DATATYPE *input;
  int input_size = batch * ic * ih * iw;

  hipError_t status = hipHostMalloc(&input, sizeof(DATATYPE) * input_size);
  init(input, input_size);

  // out is used to store the result form dev_out
  DATATYPE *out_from_dev;
  int out_size = input_size;
  hipHostMalloc(&out_from_dev, sizeof(DATATYPE) * out_size);
  memset(out_from_dev, 0, sizeof(DATATYPE) * out_size);

  DATATYPE *dev_input, *dev_out;

  // allocate the memory on the GPU
  double time1 = (double)clock() / CLOCKS_PER_SEC;
  using std::chrono::system_clock;
  system_clock::time_point today = system_clock::now();

  hipMalloc((void **)&dev_input, input_size * sizeof(DATATYPE));
  hipMalloc((void **)&dev_out, input_size * sizeof(DATATYPE));

  hipMemcpy(dev_input, input, input_size * sizeof(DATATYPE),
             hipMemcpyHostToDevice);

  for (int i = 0; i < WARMUP; i++) {
    //my_naive_nchw_nhwc(dev_input, dev_out, batch, ic, ih * iw);
    //cutlass_nchw_nhwc(dev_input, dev_out, batch, ic, ih, iw);
    //cutlass_nhwc_nchw(dev_input, dev_out, batch, ic, ih, iw);
    //hipMemcpy(dev_out, dev_input, sizeof(half) * out_size, hipMemcpyDeviceToDevice);
    //my_row_col0(dev_out, dev_input, batch, ic, ih * iw);
    //my_row_col1(dev_out, dev_input, batch, ic, ih * iw);
    cutlass_iter(dev_out, dev_input, batch, ic, ih * iw);
  }

  hipEvent_t beg, end;
  hipEventCreate(&beg);
  hipEventCreate(&end);
  hipEventRecord(beg);

  for (int i = 0; i < REPEATE; i++) {
    //my_naive_nchw_nhwc(dev_input, dev_out, batch, ic, ih * iw);
    //cutlass_nchw_nhwc(dev_input, dev_out, batch, ic, ih, iw);
    //cutlass_nhwc_nchw(dev_input, dev_out, batch, ic, ih, iw);
    // hipMemcpy(dev_out, dev_input, sizeof(half) * out_size, hipMemcpyDeviceToDevice);
    //my_row_col0(dev_out, dev_input, batch, ic, ih * iw);
    //my_row_col1(dev_out, dev_input, batch, ic, ih * iw);
    cutlass_iter(dev_out, dev_input, batch, ic, ih * iw);
  }

  hipEventRecord(end);
  hipEventSynchronize(end);
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, beg, end);
  printf("gpu layout time: %f\n", elapsed_time);

  hipMemcpy(out_from_dev, dev_out, out_size * sizeof(DATATYPE),
             hipMemcpyDeviceToHost);

  double time2 = (double)clock() / CLOCKS_PER_SEC;
  system_clock::time_point now = system_clock::now();
  auto ts = std::chrono::duration_cast<std::chrono::microseconds>(now - today);
  std::cout << "gpu total time:" << ts.count() / 1000.0 << "ms" << std::endl;
  printf("gpu total time:%lf\n", double(time2 - time1) * 1000);

  time1 = (double)clock() / CLOCKS_PER_SEC;
  today = system_clock::now();

  // results calculated in cpu is always fp32
  half *out_cpu_fp16 = (half *)malloc(sizeof(half) * out_size);
  memset(out_cpu_fp16, 0, sizeof(half) * out_size);

  naive_nchw_nhwc_cpu(input, out_cpu_fp16, batch, ic, ih, iw);
  //naive_nhwc_nchw_cpu(input, out_cpu_fp16, batch, ic, ih, iw);

  time2 = (double)clock() / CLOCKS_PER_SEC;
  now = system_clock::now();
  ts = std::chrono::duration_cast<std::chrono::microseconds>(now - today);
  std::cout << "cpu time:" << ts.count() / 1000.0 << "ms" << std::endl;
  printf("cpu time:%lf\n", double(time2 - time1) * 1000);

  printf("max_diff: %f\n", diff(out_from_dev, out_cpu_fp16, out_size));

  hipDeviceReset();
  hipHostFree(input);
  free(out_cpu_fp16);
  return 0;
}
