
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <chrono>
#include <ctime>
#include <iostream>
#include <ratio>

#include "hipblas.h"
#define FULL_MASK 0xffffffff
using DATATYPE = float;

void init(DATATYPE *a, int size) {
  for (int i = 0; i < size; i++) {
    a[i] = (rand() % 9999) / 10000.0;
  }
}

__global__ void reduce1(DATATYPE *a, int n, DATATYPE *c) {
  const int tidx = threadIdx.x;
  DATATYPE val = a[tidx];
  for (int offset = 16; offset > 0; offset /= 2) {
    val += __shfl_down_sync(FULL_MASK, val, offset);
  }
  if (tidx == 0) {
    *c = val;
  }
}

int main(void) {
  int n = 32;
  DATATYPE *a;
  hipError_t status = hipHostMalloc(&a, sizeof(DATATYPE) * n, hipHostMallocDefault);
  if (status != hipSuccess) {
    printf("分配paged内存失败");
  }
  init(a, n);

  DATATYPE c_from_gpu;

  // allocate the memory on the GPU
  DATATYPE *dev_a, *dev_c;
  hipMalloc((void **)&dev_a, n * sizeof(DATATYPE));
  hipMalloc((void **)&dev_c, sizeof(DATATYPE));
  hipMemcpy(dev_a, a, n * sizeof(DATATYPE), hipMemcpyHostToDevice);

  uint3 grid = {1, 1, 1};
  uint3 block = {32, 1, 1};

  reduce1<<<grid, block>>>(dev_a, n, dev_c);

  hipMemcpy(&c_from_gpu, dev_c, sizeof(DATATYPE), hipMemcpyDeviceToHost);

  float c_result_in_cpu = 0;
  for (int i = 0; i < n; i++) {
    c_result_in_cpu += a[i];
  }
  printf("%f\n", c_result_in_cpu);
  printf("%f\n", c_from_gpu);

  hipDeviceReset();
  hipHostFree(a);
  return 0;
}
